#include "hip/hip_runtime.h"
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "bitwisetype.h"
#include "randoms.h"
#include "rngs.h"


__global__ void setup_kernel(hiprandState* state)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	/* Each thread gets a different seed, a different sequence number, no offset */
	hiprand_init(((1237*id)%LONG_MAX),id,0,&state[id]);
}
__global__ void generate_floats(hiprandState* state, float* target, float range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandState localState = state[id];

	/* Generate pseudo-random floats*/
	target[id] = hiprand_uniform(&localState)*range;

	/* Copy state back to global memory */
	state[id] = localState;
}
__global__ void generate_ints(hiprandState* state, int* target, int range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandState localState = state[id];

	/* Generate pseudo-random ints*/
	target[id] = hiprand_uniform(&localState)*range;

	/* Copy state back to global memory */
	state[id] = localState;
}
__global__ void generate_shorts(hiprandState* state, short* target, short range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandState localState = state[id];

	/* Generate pseudo-random shorts */
	target[id] = hiprand_uniform(&localState)*range;

	/* Copy state back to global memory */
	state[id] = localState;
}

__global__ void generate_bits(hiprandState* state, BitWiseType* target)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandState localState = state[id];
	BitWiseType* temp;
	short ts;

	/* Generate pseudo-random shorts */
	ts = (hiprand_uniform(&localState)*32768);
	temp = reinterpret_cast <BitWiseType*> (&ts);
	target[id] = *temp;

	/* Copy state back to global memory */
	state[id] = localState;
}


