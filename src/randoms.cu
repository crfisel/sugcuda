#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "randoms.h"

__global__ void setup_kernel(hiprandStateXORWOW_t* state)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	/* Each thread gets same seed, unique sequence number, no offset */
	hiprand_init(1234567,id,0,&state[id]);
}
__global__ void generate_floats(hiprandStateXORWOW_t* state, float* target, float range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandStateXORWOW_t localState = state[id];
	float fTemp = hiprand_uniform(&localState)*range;
	
	/* Generate pseudo-random floats*/
	target[id] = fTemp;

	/* Copy state back to global memory */
	state[id] = localState;
}
__global__ void generate_ints(hiprandStateXORWOW_t* state, int* target, int range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandStateXORWOW_t localState = state[id];
	float fTemp = hiprand_uniform(&localState)*range;

	/* Generate pseudo-random ints*/
	target[id] = fTemp;

	/* Copy state back to global memory */
	state[id] = localState;
}
__global__ void generate_shorts(hiprandStateXORWOW_t* state, short* target, short range)
{
	/* Copy to local memory for efficiency */
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprandStateXORWOW_t localState = state[id];
	float fTemp = hiprand_uniform(&localState)*range;

	/* Generate pseudo-random shorts */
	target[id] = fTemp;

	/* Copy state back to global memory */
	state[id] = localState;
}

