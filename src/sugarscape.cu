/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include "constants.h"
#include "randoms.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "exercise_locks.h"
#include "grow_back1.h"

int main (int argc , char* argv [])
{
	// use the GTX470
	CUDA_CALL(hipSetDevice(0));

	int ihMaxBlocks = (MAX_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
	int ihInitBlocks = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// set up grid-oriented states
	hiprandState* pgStates = 0;
	CUDA_CALL(hipMalloc((void **)&pgStates,GRID_SIZE*GRID_SIZE*sizeof(hiprandState)));
	setup_kernel<<<GRID_SIZE,GRID_SIZE>>>(pgStates);

	// set up agent-oriented states
	hiprandState* paStates = 0;
	CUDA_CALL(hipMalloc((void **)&paStates,2*MAX_AGENTS*sizeof(hiprandState)));
	setup_kernel<<<ihMaxBlocks,NUM_THREADS_PER_BLOCK>>>(paStates);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,MAX_AGENTS*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,MAX_AGENTS*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));

	// and its host-side copy
	int* pihDeferredQueueSize;
	CUDA_CALL(hipHostAlloc((void**)&pihDeferredQueueSize,sizeof(int),hipHostMallocDefault));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));

	// and its host-side copy
	int* pihLockSuccesses;
	CUDA_CALL(hipHostAlloc((void**)&pihLockSuccesses,sizeof(int),hipHostMallocDefault));

	// the (dynamic) population counter
	int* piPopulation;
	CUDA_CALL(hipMalloc((void**)&piPopulation,sizeof(int)));

	// and its host-side copy
	int* pihPopulation;
	CUDA_CALL(hipHostAlloc((void**)&pihPopulation,sizeof(int),hipHostMallocDefault));

	// the static agents counter
	int* piStaticAgents;
	CUDA_CALL(hipMalloc((void**)&piStaticAgents,sizeof(int)));

	// and its host-side copy
	int* pihStaticAgents;
	CUDA_CALL(hipHostAlloc((void**)&pihStaticAgents,sizeof(int),hipHostMallocDefault));

	// initialize agent properties
	// position in x and y
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,MAX_AGENTS*sizeof(short)));
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,MAX_AGENTS*sizeof(short)));

	// bit storage
	int* piaBits;
	CUDA_CALL(hipMalloc((void**)&piaBits,MAX_AGENTS*sizeof(int)));

	// sugar and spice holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,MAX_AGENTS*sizeof(float)));
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,MAX_AGENTS*sizeof(float)));

	// initial sugar and spice holdings
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,MAX_AGENTS*sizeof(float)));
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,MAX_AGENTS*sizeof(float)));

	// initialize grid properties
	// sugar and spice in square
	int* pigBits;
	CUDA_CALL(hipMalloc((void**)&pigBits,GRID_SIZE*GRID_SIZE*sizeof(int)));

	pihPopulation[0] = INIT_AGENTS;
	CUDA_CALL(hipMemcpy(piPopulation,pihPopulation,sizeof(int),hipMemcpyHostToDevice));

	CUDA_CALL(hipDeviceSynchronize());

	fill_positions<<<ihInitBlocks,NUM_THREADS_PER_BLOCK>>>(paStates,psaX,psaY,GRID_SIZE);

	initialize_agentbits<<<ihInitBlocks,NUM_THREADS_PER_BLOCK>>>(paStates,piaBits);

	initialize_food<<<ihInitBlocks,NUM_THREADS_PER_BLOCK>>>(pfaSugar,pfaSpice,paStates,30.0f);
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(pgStates,pigBits,TILED);

	CUDA_CALL(hipDeviceSynchronize());

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&end));
	CUDA_CALL(hipEventRecord(start,0));

	// count occupancy and store residents

	int status = exercise_locks(COUNT,psaX,psaY,piaBits,pigBits,pigResidents,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,piaQueueA,piaQueueB,
			piPopulation,pihPopulation,piDeferredQueueSize,pihDeferredQueueSize,paStates,piLockSuccesses,pihLockSuccesses,piStaticAgents,pihStaticAgents);

	//   end timing
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventRecord(end,0));
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

	printf ("Counting %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// main loop
	while(pihPopulation[0] > 10) {

		int ihCurrBlocks = (pihPopulation[0]+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

		// time movement
		CUDA_CALL(hipEventRecord(start,0));

		// do movement
		status = exercise_locks(MOVE,psaX,psaY,piaBits,pigBits,pigResidents,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,piaQueueA,piaQueueB,
				piPopulation,pihPopulation,piDeferredQueueSize,pihDeferredQueueSize,paStates,piLockSuccesses,pihLockSuccesses,piStaticAgents,pihStaticAgents);
		CUDA_CALL(hipDeviceSynchronize());

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		printf ("Moving %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time harvest, meal and aging
		CUDA_CALL(hipEventRecord(start,0));

		harvest<<<GRID_SIZE,GRID_SIZE>>>(psaX,pigBits,pigResidents,pfaSugar,pfaSpice,pgStates);

		CUDA_CALL(hipDeviceSynchronize());

		eat<<<ihCurrBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaBits,pfaSugar,pfaSpice);

		age<<<ihCurrBlocks,NUM_THREADS_PER_BLOCK>>>(piaBits);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		printf ("Harvesting %d squares and feeding and aging %d agents takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) pihPopulation[0], (float) elapsed_time);

/*		// time mating
		CUDA_CALL(hipEventRecord(start,0));

		status = exercise_locks(MATE,psaX,psaY,piaBits,pigBits,pigResidents,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,piaQueueA,piaQueueB,
				piPopulation,pihPopulation,piDeferredQueueSize,pihDeferredQueueSize,paStates,piLockSuccesses,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Mating %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);
*/

		// time dying
		CUDA_CALL(hipEventRecord(start,0));

		status = exercise_locks(DIE,psaX,psaY,piaBits,pigBits,pigResidents,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,piaQueueA,piaQueueB,
				piPopulation,pihPopulation,piDeferredQueueSize,pihDeferredQueueSize,paStates,piLockSuccesses,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Registering deaths among %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time regrowth
		CUDA_CALL(hipEventRecord(start,0));

		grow_back1<<<GRID_SIZE,GRID_SIZE>>>(pigBits);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));
		CUDA_CALL(hipDeviceSynchronize());
		printf ("Growing sugar and spice on %d squares takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (float) elapsed_time);
	}

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipHostFree(pihLockSuccesses));
	CUDA_CALL(hipHostFree(pihDeferredQueueSize));
	CUDA_CALL(hipHostFree(pihStaticAgents));
	CUDA_CALL(hipFree(paStates));
	CUDA_CALL(hipFree(pgStates));

	return EXIT_SUCCESS;
}
