/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include "symbolic_constants.h"
#include "randoms.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "exercise_locks.h"
#include "grow_back1.h"
#include "rngs.h"

int main (int argc , char* argv [])
{
	// use the GTX470
	hipSetDevice(0);

    unsigned int* piaRandoms;
    unsigned int* pigRandoms;
    unsigned int* piahTemp;
    float* pfahTemp;
    unsigned int* pighTemp;

	// seed rngs
	SelectStream(0);
	PutSeed(1234567);

	// initialize agent properties
	int hNumBlocks = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
	CUDA_CALL(hipMalloc((void**)&piaRandoms,MAX_AGENTS*sizeof(int)));

	// position in x and y
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,MAX_AGENTS*sizeof(short)));
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,MAX_AGENTS*sizeof(short)));
	CUDA_CALL(hipHostAlloc((void**)&piahTemp,MAX_AGENTS*sizeof(int),hipHostMallocDefault));
	for (int i = 0; i < INIT_AGENTS; i++) {
		piahTemp[i] = Random()*GRID_SIZE*(GRID_SIZE-0.01);
	}
	CUDA_CALL(hipMemcpy(piaRandoms,piahTemp,INIT_AGENTS*sizeof(int),hipMemcpyHostToDevice));
	fill_positions<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,psaX,psaY);	

	// bit storage
	int* piaAgentBits;
	CUDA_CALL(hipMalloc((void**)&piaAgentBits,MAX_AGENTS*sizeof(int)));
	for (int i = 0; i < INIT_AGENTS; i++) {
		piahTemp[i] = Random_uint();
	}
	CUDA_CALL(hipMemcpy(piaRandoms,piahTemp,INIT_AGENTS*sizeof(int),hipMemcpyHostToDevice));
	initialize_agentbits<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,piaAgentBits);
	
	// sugar holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipHostAlloc((void**)&pfahTemp,MAX_AGENTS*sizeof(float),hipHostMallocDefault));
		for (int i = 0; i < INIT_AGENTS; i++) {
		pfahTemp[i] = Random()*30.0f;
	}
	CUDA_CALL(hipMemcpy(pfaSugar,pfahTemp,INIT_AGENTS*sizeof(int),hipMemcpyHostToDevice));
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// spice holdings
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,MAX_AGENTS*sizeof(float)));
	for (int i = 0; i < INIT_AGENTS; i++) {
		pfahTemp[i] = Random()*30.0f;
	}
	CUDA_CALL(hipMemcpy(pfaSpice,pfahTemp,INIT_AGENTS*sizeof(int),hipMemcpyHostToDevice));
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// initialize grid properties
	CUDA_CALL(hipMalloc((void**)&pigRandoms,GRID_SIZE*GRID_SIZE*sizeof(int)));

	// sugar in square
	int* pigGridBits;
	CUDA_CALL(hipMalloc((void**)&pigGridBits,GRID_SIZE*GRID_SIZE*sizeof(int)));
	CUDA_CALL(hipHostAlloc((void**)&pighTemp,GRID_SIZE*GRID_SIZE*sizeof(int),hipHostMallocDefault));
	for (int i = 0; i < GRID_SIZE*GRID_SIZE; i++) {
		pighTemp[i] = Random()*UINT_MAX;
	}
	CUDA_CALL(hipMemcpy(pigRandoms,pighTemp,GRID_SIZE*GRID_SIZE*sizeof(int),hipMemcpyHostToDevice));
	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(pigRandoms,pigGridBits,TILED);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,MAX_AGENTS*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,MAX_AGENTS*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));
	
	// and its host-side copy
	int* pihDeferredQueueSize;
	CUDA_CALL(hipHostAlloc((void**)&pihDeferredQueueSize,sizeof(int),hipHostMallocDefault));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));
	
	// and its host-side copy
	int* pihLockSuccesses;
	CUDA_CALL(hipHostAlloc((void**)&pihLockSuccesses,sizeof(int),hipHostMallocDefault));

	// the (dynamic) population counter
	int* piPopulation;
	CUDA_CALL(hipMalloc((void**)&piPopulation,sizeof(int)));

	// and its host-side copy
	int* pihPopulation;
	CUDA_CALL(hipHostAlloc((void**)&pihPopulation,sizeof(int),hipHostMallocDefault));
	pihPopulation[0] = INIT_AGENTS;
	CUDA_CALL(hipMemcpy(piPopulation,pihPopulation,sizeof(int),hipMemcpyHostToDevice));

	// the static agents counter
	int* piStaticAgents;
	CUDA_CALL(hipMalloc((void**)&piStaticAgents,sizeof(int)));
	
	// and its host-side copy
	int* pihStaticAgents;
	CUDA_CALL(hipHostAlloc((void**)&pihStaticAgents,sizeof(int),hipHostMallocDefault));
	
	CUDA_CALL(hipDeviceSynchronize());

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&end));
	CUDA_CALL(hipEventRecord(start,0));

	// count occupancy and store residents

	int status = exercise_locks(COUNT,psaX,psaY,piaAgentBits,piaRandoms,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
		piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

	//   end timing
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventRecord(end,0));
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));
	
	printf ("Counting %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);

	// main loop
		while(pihPopulation[0] > 10) {
		// time movement
		CUDA_CALL(hipEventRecord(start,0));

		// do movement
		status = exercise_locks(MOVE,psaX,psaY,piaAgentBits,piaRandoms,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);
		CUDA_CALL(hipDeviceSynchronize());

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		printf ("Moving %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);

		for (int i = 0; i < GRID_SIZE*GRID_SIZE; i++) {
			pighTemp[i] = Random()*UINT_MAX;
		}
		CUDA_CALL(hipMemcpy(pigRandoms,pighTemp,GRID_SIZE*GRID_SIZE*sizeof(int),hipMemcpyHostToDevice));
		harvest<<<GRID_SIZE,GRID_SIZE>>>(pigRandoms,psaX,pfaSugar,pfaSpice,pigGridBits,pigResidents);
	
		eat<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaAgentBits,pfaSugar,pfaSpice);

		age<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaAgentBits);

		// time mating
		CUDA_CALL(hipEventRecord(start,0));

		for (int i = 0; i < pihPopulation[0]; i++) {
			piahTemp[i] = Random()*UINT_MAX;
		}
		CUDA_CALL(hipMemcpy(piaRandoms,piahTemp,pihPopulation[0]*sizeof(int),hipMemcpyHostToDevice));
		status = exercise_locks(MATE,psaX,psaY,piaAgentBits,piaRandoms,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Mating %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);

		// time dying
		CUDA_CALL(hipEventRecord(start,0));

		status = exercise_locks(DIE,psaX,psaY,piaAgentBits,piaRandoms,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Registering deaths among %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);

		grow_back1<<<GRID_SIZE,GRID_SIZE>>>(pigGridBits);
	}

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaAgentBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigGridBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipFree(piaRandoms));
	CUDA_CALL(hipFree(pigRandoms));
	CUDA_CALL(hipHostFree(pihLockSuccesses));
	CUDA_CALL(hipHostFree(pihDeferredQueueSize));
	CUDA_CALL(hipHostFree(pihStaticAgents));
	CUDA_CALL(hipHostFree(piahTemp));
	CUDA_CALL(hipHostFree(pfahTemp));
	CUDA_CALL(hipHostFree(pighTemp));

	return EXIT_SUCCESS;
}
