/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include "symbolic_constants.h"
#include "randoms.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "exercise_locks.h"
#include "grow_back1.h"

int main (int argc , char* argv [])
{
	// use the GTX470
	CUDA_CALL(hipSetDevice(0));

	int hMaxBlocks = (MAX_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
	int hInitBlocks = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// set up grid-oriented states
	hiprandState* pgStates = 0;
	CUDA_CALL(hipMalloc((void **)&pgStates,GRID_SIZE*GRID_SIZE*sizeof(hiprandState)));
	setup_kernel<<<GRID_SIZE,GRID_SIZE>>>(pgStates);

	// set up agent-oriented states
	hiprandState* paStates = 0;
	CUDA_CALL(hipMalloc((void **)&paStates,2*MAX_AGENTS*sizeof(hiprandState)));
	setup_kernel<<<hMaxBlocks,NUM_THREADS_PER_BLOCK>>>(paStates);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,MAX_AGENTS*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,MAX_AGENTS*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));

	// and its host-side copy
	int* pihDeferredQueueSize;
	CUDA_CALL(hipHostAlloc((void**)&pihDeferredQueueSize,sizeof(int),hipHostMallocDefault));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));

	// and its host-side copy
	int* pihLockSuccesses;
	CUDA_CALL(hipHostAlloc((void**)&pihLockSuccesses,sizeof(int),hipHostMallocDefault));

	// the (dynamic) population counter
	int* piPopulation;
	CUDA_CALL(hipMalloc((void**)&piPopulation,sizeof(int)));

	// and its host-side copy
	int* pihPopulation;
	CUDA_CALL(hipHostAlloc((void**)&pihPopulation,sizeof(int),hipHostMallocDefault));

	// the static agents counter
	int* piStaticAgents;
	CUDA_CALL(hipMalloc((void**)&piStaticAgents,sizeof(int)));

	// and its host-side copy
	int* pihStaticAgents;
	CUDA_CALL(hipHostAlloc((void**)&pihStaticAgents,sizeof(int),hipHostMallocDefault));

	// initialize agent properties
	// position in x and y
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,MAX_AGENTS*sizeof(short)));
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,MAX_AGENTS*sizeof(short)));

	// bit storage
	int* piaAgentBits;
	CUDA_CALL(hipMalloc((void**)&piaAgentBits,MAX_AGENTS*sizeof(int)));

	// sugar and spice holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,MAX_AGENTS*sizeof(float)));
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,MAX_AGENTS*sizeof(float)));

	// initial sugar and spice holdings
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,MAX_AGENTS*sizeof(float)));
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,MAX_AGENTS*sizeof(float)));

	// initialize grid properties
	// sugar and spice in square
	int* pigGridBits;
	CUDA_CALL(hipMalloc((void**)&pigGridBits,GRID_SIZE*GRID_SIZE*sizeof(int)));

	pihPopulation[0] = INIT_AGENTS;
	CUDA_CALL(hipMemcpy(piPopulation,pihPopulation,sizeof(int),hipMemcpyHostToDevice));

	CUDA_CALL(hipDeviceSynchronize());

	fill_positions<<<hInitBlocks,NUM_THREADS_PER_BLOCK>>>(paStates,psaX,psaY,GRID_SIZE);

	initialize_agentbits<<<hInitBlocks,NUM_THREADS_PER_BLOCK>>>(paStates,piaAgentBits);

	initialize_food<<<hInitBlocks,NUM_THREADS_PER_BLOCK>>>(pfaSugar,pfaSpice,paStates,30.0f);
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(pgStates,pigGridBits,TILED);

	CUDA_CALL(hipDeviceSynchronize());

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&end));
	CUDA_CALL(hipEventRecord(start,0));

	// count occupancy and store residents

	int status = exercise_locks(COUNT,paStates,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

	//   end timing
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventRecord(end,0));
	CUDA_CALL(hipEventSynchronize(end));
	CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

	printf ("Counting %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// main loop
	while(pihPopulation[0] > 10) {;
		// time movement
		CUDA_CALL(hipEventRecord(start,0));

		// do movement
		status = exercise_locks(MOVE,paStates,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
				piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);
		CUDA_CALL(hipDeviceSynchronize());

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		printf ("Moving %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time harvest, meal and aging
		CUDA_CALL(hipEventRecord(start,0));

		harvest<<<GRID_SIZE,GRID_SIZE>>>(pgStates,psaX,pfaSugar,pfaSpice,pigGridBits,pigResidents);

		eat<<<hInitBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaAgentBits,pfaSugar,pfaSpice);

		age<<<hInitBlocks,NUM_THREADS_PER_BLOCK>>>(piaAgentBits);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		printf ("Harvesting %d squares and feeding and aging %d agents takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) pihPopulation[0], (float) elapsed_time);

		// time mating
		CUDA_CALL(hipEventRecord(start,0));

		status = exercise_locks(MATE,paStates,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Mating %d agents takes %f ms\n",(int) pihPopulation[0], (float) elapsed_time);


		// time dying
		CUDA_CALL(hipEventRecord(start,0));

		status = exercise_locks(DIE,paStates,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pfaInitialSugar,pfaInitialSpice,pigGridBits,pigResidents,piaQueueA,
				piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses,piStaticAgents,pihStaticAgents);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Registering deaths among %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time regrowth
		CUDA_CALL(hipEventRecord(start,0));

		grow_back1<<<GRID_SIZE,GRID_SIZE>>>(pigGridBits);

		//   end timing
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventRecord(end,0));
		CUDA_CALL(hipEventSynchronize(end));
		CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));
		CUDA_CALL(hipDeviceSynchronize());
		printf ("Growing sugar and spice on %d squares takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (float) elapsed_time);
	}

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaAgentBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigGridBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipHostFree(pihLockSuccesses));
	CUDA_CALL(hipHostFree(pihDeferredQueueSize));
	CUDA_CALL(hipHostFree(pihStaticAgents));
	CUDA_CALL(hipFree(paStates));
	CUDA_CALL(hipFree(pgStates));

	return EXIT_SUCCESS;
}
