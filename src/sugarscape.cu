/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include "symbolic_constants.h"
#include "common_config.h"
#include "cudpp.h"
#include "randoms.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "exercise_locks.h"
#include "grow_back1.h"
#include "aggregate.h"

enum ReduceType
{
    REDUCE_INT,
    REDUCE_FLOAT,
    REDUCE_DOUBLE
};

#define MAX_BLOCK_DIM_SIZE 65535

extern "C"
bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

int main (int argc , char* argv [])
{
	// use the GTX470
	hipSetDevice(0);

    unsigned int seed = 9999;   //constant seed
    unsigned int* piaRandoms;
    unsigned int* pigRandoms;

    //initialize the CUDPP config
    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_RAND_MD5;
    config.options = 0;

    CUDPPHandle randPlan = 0;
    CUDPPResult result;

    CUDPPHandle theCudpp;
    result = cudppCreate(&theCudpp);
	if(result != CUDPP_SUCCESS)
	{
		printf("Error initializing CUDPP Library.\n");
		return -1;
	}

	CUDA_CALL(hipMalloc((void**)&piaRandoms,MAX_AGENTS*sizeof(unsigned int)));
	CUDA_CALL(hipMalloc((void**)&pigRandoms,MAX_AGENTS*sizeof(unsigned int)));
	result = cudppPlan(theCudpp,&randPlan,config,MAX_AGENTS,1,0);

	if (CUDPP_SUCCESS != result)
        {
            printf("Error creating CUDPPPlan\n");
            exit(-1);
        }

	cudppRandSeed(randPlan, seed);
	cudppRand(randPlan,piaRandoms,INIT_AGENTS);

// initialize agent properties
	// setup dimensions
	int hNumBlocks = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// position in x and y
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,MAX_AGENTS*sizeof(short)));
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,MAX_AGENTS*sizeof(short)));
	fill_positions<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,psaX,psaY);	

	// create agent arrays on device
	// bit storage
	int* piaAgentBits;
	CUDA_CALL(hipMalloc((void**)&piaAgentBits,MAX_AGENTS*sizeof(int)));
	cudppRand(randPlan,piaRandoms,INIT_AGENTS);
	initialize_agentbits<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,piaAgentBits);
	
	// sugar holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,MAX_AGENTS*sizeof(float)));
	cudppRand(randPlan,piaRandoms,INIT_AGENTS);
	initialize_food<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,pfaSugar,30.0f);	
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// spice holdings
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,MAX_AGENTS*sizeof(float)));
	cudppRand(randPlan,piaRandoms,INIT_AGENTS);
	initialize_food<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaRandoms,pfaSpice,30.0f);	
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// create grid on device
	// sugar in square
	int* pigGridBits;
	CUDA_CALL(hipMalloc((void**)&pigGridBits,GRID_SIZE*GRID_SIZE*sizeof(int)));
	cudppRand(randPlan,piaRandoms,GRID_SIZE*GRID_SIZE);
	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(pigRandoms,pigGridBits);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,MAX_AGENTS*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,MAX_AGENTS*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));
	
	// and its host-side copy
	int* pihDeferredQueueSize = (int*) malloc(sizeof(int));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));
	
	// and its host-side copy
	int* pihLockSuccesses = (int*) malloc(sizeof(int));
	
	// the (dynamic) population counter
	int* piPopulation;
	CUDA_CALL(hipMalloc((void**)&piPopulation,sizeof(int)));
	int* pihPopulation = (int*) malloc(sizeof(int)); 
	pihPopulation[0] = INIT_AGENTS;
	CUDA_CALL(hipMemcpy(piPopulation,pihPopulation,sizeof(int),hipMemcpyHostToDevice));

	hipDeviceSynchronize();

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start,0);

	// count occupancy and store residents

	int status = exercise_locks(COUNT,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pigGridBits,pigResidents,piaQueueA,
		piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	
	printf ("Counting %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// main loop
		while(pihPopulation[0] > 10) {
		// time movement
		hipEventRecord(start,0);

		// do movement
		status = exercise_locks(MOVE,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses);
		hipDeviceSynchronize();

		//   end timing
		hipDeviceSynchronize();
		hipEventSynchronize(end);
		hipEventRecord(end,0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&elapsed_time, start, end);

		printf ("Moving %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time harvest, meal and aging
		hipEventRecord(start,0);

		cudppRand(randPlan,pigRandoms,GRID_SIZE*GRID_SIZE);
		harvest<<<GRID_SIZE,GRID_SIZE>>>(pigRandoms,psaX,pfaSugar,pfaSpice,pigGridBits,pigResidents);
	
		eat<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaAgentBits,pfaSugar,pfaSpice);

		age<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(piaAgentBits);

		//   end timing
		hipDeviceSynchronize();
		hipEventSynchronize(end);
		hipEventRecord(end,0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&elapsed_time, start, end);

		printf ("Harvesting %d squares and feeding and aging %d agents takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) pihPopulation[0], (float) elapsed_time);

		// time dying
		hipEventRecord(start,0);

		status = exercise_locks(DIE,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pigGridBits,pigResidents,piaQueueA,
			piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses,pihDeferredQueueSize,pihLockSuccesses);

		//   end timing
		hipDeviceSynchronize();
		hipEventSynchronize(end);
		hipEventRecord(end,0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&elapsed_time, start, end);

		CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
		printf ("Registering deaths among %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

		// time regrowth
		hipEventRecord(start,0);

		grow_back1<<<GRID_SIZE,GRID_SIZE>>>(pigGridBits);

		//   end timing
		hipDeviceSynchronize();
		hipEventSynchronize(end);
		hipEventRecord(end,0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&elapsed_time, start, end);
		hipDeviceSynchronize();
		printf ("Growing sugar and spice on %d squares takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (float) elapsed_time);
	}

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaAgentBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigGridBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	free(pihLockSuccesses);
	free(pihDeferredQueueSize);

	CUDA_CALL(hipFree(piaRandoms));
	CUDA_CALL(hipFree(pigRandoms));

	result = cudppDestroyPlan(randPlan);
	if (CUDPP_SUCCESS != result) {
			printf("Error destroying CUDPPPlan\n");
			exit(-1);
	}
	
	result = cudppDestroy(theCudpp);
	if (CUDPP_SUCCESS != result) {
		printf("Error shutting down CUDPP Library.\n");
		exit(-1);
	}

	return EXIT_SUCCESS;
}
