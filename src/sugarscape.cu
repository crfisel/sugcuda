/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "count.h"
#include "randoms.h"
#include "move.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "die.h"

int main (int argc , char* argv [])
{
	hiprandStateXORWOW_t* devAgentStates = 0;
	hiprandStateXORWOW_t* devGridStates = 0;

	// use the GTX470
	hipSetDevice(0);

	// Allocate and set up prng states on device
	CUDA_CALL(hipMalloc((void**)&devAgentStates,NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK*sizeof(hiprandStateXORWOW_t)));
	CUDA_CALL(hipMalloc((void**)&devGridStates,GRID_SIZE*GRID_SIZE*sizeof(hiprandStateXORWOW_t)));
	setup_kernel<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(devAgentStates);
	setup_kernel<<<GRID_SIZE,GRID_SIZE>>>(devGridStates);

	// create agent arrays on device
	int hPopulation = NUM_AGENTS;

	// position in x
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,hPopulation*sizeof(short)));

	//position in y
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,hPopulation*sizeof(short)));

	// bit storage
	int* piaAgentBits;
	CUDA_CALL(hipMalloc((void**)&piaAgentBits,hPopulation*sizeof(int)));

	// sugar holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,hPopulation*sizeof(float)));

	// spice holdings
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,hPopulation*sizeof(float)));

	// current age
	short* psaAge;
	CUDA_CALL(hipMalloc((void**)&psaAge,hPopulation*sizeof(short)));

	// initialize agent properties
	// setup dimensions
	int hNumTiles = (hPopulation+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK/NUM_THREADS_PER_BLOCK;
	int hNumBlocks = (hPopulation+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// for large numbers of agents, tile the prngs
	if (hPopulation > NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK) {
		for (int i = 0; i < hNumTiles; i++) {
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaX[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE-1));
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaY[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE-1));
			generate_ints<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(piaAgentBits[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),INT_MAX);
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSugar[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),30.0f);
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSpice[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),30.0f);
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaAge[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),100);
		}
	} else {
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaX,GRID_SIZE-1);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaY,GRID_SIZE-1);
		generate_ints<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,piaAgentBits,INT_MAX);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSugar,30.0f);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSpice,30.0f);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaAge,100);
	}

	// initial sugar holdings
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,hPopulation*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,hPopulation*sizeof(float),hipMemcpyDeviceToDevice));

	// initial spice holdings
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,hPopulation*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,hPopulation*sizeof(float),hipMemcpyDeviceToDevice));

	// create grid on device
	// sugar in square
	int* pigGridBits;
	CUDA_CALL(hipMalloc((void**)&pigGridBits,GRID_SIZE*GRID_SIZE*sizeof(int)));

	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,pigGridBits);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,hPopulation*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,hPopulation*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));
	
	hipDeviceSynchronize();

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start,0);

	// count occupancy and store residents
	int status = count(psaX,psaY,pigGridBits,pigResidents,piaQueueA,hPopulation,
		piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Counting %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time movement
	hipEventRecord(start,0);

	// do movement
	move(psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,pigGridBits,pigResidents,
		piaQueueA,hPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);
	hipDeviceSynchronize();

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Moving %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time harvest
	hipEventRecord(start,0);

	harvest<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,psaX,pfaSugar,pfaSpice,pigGridBits,pigResidents);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Harvesting %d grid squares takes %d milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) elapsed_time);

	// time meal
	hipEventRecord(start,0);

	eat<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaAgentBits,pfaSugar,pfaSpice);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Feeding %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time aging
	hipEventRecord(start,0);

	age<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaAge);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);

	printf ("Aging %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time dying
	hipEventRecord(start,0);

	die(psaX,psaY,piaAgentBits,psaAge,pfaSugar,pfaSpice,pigGridBits,pigResidents,
		piaQueueA,hPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Registering deaths among %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaAgentBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(psaAge));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigGridBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipFree(devAgentStates));
	CUDA_CALL(hipFree(devGridStates));

	return EXIT_SUCCESS ;
}
