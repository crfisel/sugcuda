/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "count.h"
#include "randoms.h"
#include "move.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "die.h"

int main (int argc , char* argv [])
{
	hiprandStateXORWOW_t* devAgentStates = 0;
	hiprandStateXORWOW_t* devGridStates = 0;

	// use the GTX470
	hipSetDevice(0);

	// Allocate and set up prng states on device
	CUDA_CALL(hipMalloc((void**)&devAgentStates,NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK*sizeof(hiprandStateXORWOW_t)));
	CUDA_CALL(hipMalloc((void**)&devGridStates,GRID_SIZE*GRID_SIZE*sizeof(hiprandStateXORWOW_t)));
	setup_kernel<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(devAgentStates);
	setup_kernel<<<GRID_SIZE,GRID_SIZE>>>(devGridStates);

	// create agent arrays on device
	int hPopulation = NUM_AGENTS;

	// position in x
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,hPopulation*sizeof(short)));

	//position in y
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,hPopulation*sizeof(short)));

	// bit storage
	int* piaBits;
	CUDA_CALL(hipMalloc((void**)&piaBits,hPopulation*sizeof(int)));

	// sugar holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,hPopulation*sizeof(float)));

	// spice holdings
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,hPopulation*sizeof(float)));

	// current age
	short* psaAge;
	CUDA_CALL(hipMalloc((void**)&psaAge,hPopulation*sizeof(short)));

	// initialize agent properties
	// setup dimensions
	int hNumTiles = (hPopulation+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK/NUM_THREADS_PER_BLOCK;
	int hNumBlocks = (hPopulation+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// for large numbers of agents, tile the prngs
	if (hPopulation > NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK) {
		for (int i = 0; i < hNumTiles; i++) {
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaX[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE-1));
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaY[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE-1));
			generate_ints<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(piaBits[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),INT_MAX);
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSugar[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),4.0f);
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSpice[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),4.0f);
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaAge[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),100);
		}
	} else {
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaX,GRID_SIZE-1);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaY,GRID_SIZE-1);
		generate_ints<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,piaBits,INT_MAX);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSugar,4.0f);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSpice,4.0f);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaAge,100);
	}

	// initial sugar holdings
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,hPopulation*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,hPopulation*sizeof(float),hipMemcpyDeviceToDevice));

	// initial spice holdings
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,hPopulation*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,hPopulation*sizeof(float),hipMemcpyDeviceToDevice));

	// create grid on device
	// sugar in square
	short* psgSugar;
	CUDA_CALL(hipMalloc((void**)&psgSugar,GRID_SIZE*GRID_SIZE*sizeof(short)));

	// spice in square
	short* psgSpice;
	CUDA_CALL(hipMalloc((void**)&psgSpice,GRID_SIZE*GRID_SIZE*sizeof(short)));

	generate_shorts<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,psgSugar,4);
	generate_shorts<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,psgSpice,4);

	// occupancy of square - initially zero
	short* psgOccupancy;
	CUDA_CALL(hipMalloc((void**)&psgOccupancy,GRID_SIZE*GRID_SIZE*sizeof(short)));
	CUDA_CALL(hipMemset(psgOccupancy,0,GRID_SIZE*GRID_SIZE*sizeof(short)));

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// provision for locking the square - set to unlocked
	int* pigLocks;
	CUDA_CALL(hipMalloc((void**)&pigLocks,GRID_SIZE*GRID_SIZE*sizeof(int)));
	CUDA_CALL(hipMemset(pigLocks,0,GRID_SIZE*GRID_SIZE*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,hPopulation*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,hPopulation*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));
	
	hipDeviceSynchronize();

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start,0);

	// count occupancy and store residents
	int status = count(psaX,psaY,psgOccupancy,pigResidents,pigLocks,piaQueueA,hPopulation,
		piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Counting %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time movement
	hipEventRecord(start,0);

	// do movement
	move(psaX,psaY,piaBits,pfaSugar,pfaSpice,psgSugar,psgSpice,psgOccupancy,pigResidents,
		pigLocks,piaQueueA,hPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);
	hipDeviceSynchronize();

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Moving %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time harvest
	hipEventRecord(start,0);

	harvest<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,psaX,pfaSugar,pfaSpice,psgSugar,psgSpice,
			psgOccupancy,pigResidents);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Harvesting %d grid squares takes %d milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) elapsed_time);

	// time meal
	hipEventRecord(start,0);

	eat<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaBits,pfaSugar,pfaSpice);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Feeding %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time aging
	hipEventRecord(start,0);

	age<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaAge);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);

	printf ("Aging %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// time dying
	hipEventRecord(start,0);

	die(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,psgOccupancy, 
			pigResidents,pigLocks,piaQueueA,hPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	printf ("Registering deaths among %d agents takes %d milliseconds\n",(int) hPopulation, (int) elapsed_time);

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(psaAge));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(psgSugar));
	CUDA_CALL(hipFree(psgSpice));
	CUDA_CALL(hipFree(psgOccupancy));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(pigLocks));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipFree(devAgentStates));
	CUDA_CALL(hipFree(devGridStates));
	return EXIT_SUCCESS ;
}
