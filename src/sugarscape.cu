/* 
 *  sugarscape.cu
 *
 * *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "rngs.h"
#include "randoms.h"
#include "harvest.h"
#include "eat.h"
#include "age.h"
#include "exercise_locks.h"

int main (int argc , char* argv [])
{
	hiprandStateXORWOW_t* devAgentStates = 0;
	hiprandStateXORWOW_t* devGridStates = 0;

	// use the GTX470
	hipSetDevice(0);

	// Allocate and set up prng states on device
	CUDA_CALL(hipMalloc((void**)&devAgentStates,NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK*sizeof(hiprandStateXORWOW_t)));
	CUDA_CALL(hipMalloc((void**)&devGridStates,GRID_SIZE*GRID_SIZE*sizeof(hiprandStateXORWOW_t)));
	setup_kernel<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(devAgentStates);
	setup_kernel<<<GRID_SIZE,GRID_SIZE>>>(devGridStates);

	// create agent arrays on device
	// bit storage
	int* piaAgentBits;
	CUDA_CALL(hipMalloc((void**)&piaAgentBits,MAX_AGENTS*sizeof(int)));

	// sugar holdings
	float* pfaSugar;
	CUDA_CALL(hipMalloc((void**)&pfaSugar,MAX_AGENTS*sizeof(float)));

	// spice holdings
	float* pfaSpice;
	CUDA_CALL(hipMalloc((void**)&pfaSpice,MAX_AGENTS*sizeof(float)));

	// current age
	short* psaAge;
	CUDA_CALL(hipMalloc((void**)&psaAge,MAX_AGENTS*sizeof(short)));

	// initialize agent properties
	// setup dimensions
	int hNumTiles = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK/NUM_THREADS_PER_BLOCK;
	int hNumBlocks = (INIT_AGENTS+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;

	// for large numbers of agents, tile the prngs
	if (INIT_AGENTS > NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK) {
		for (int i = 0; i < hNumTiles; i++) {
/*			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaX[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE));
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaY[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),(GRID_SIZE));
*/			generate_ints<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(piaAgentBits[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]));
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSugar[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),30.0f);
			generate_floats<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(pfaSpice[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),30.0f);
			generate_shorts<<<NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(
					devAgentStates,&(psaAge[i*NUM_THREADS_PER_BLOCK*NUM_THREADS_PER_BLOCK]),100);
		}
	} else {
/*		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaX,GRID_SIZE);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaY,GRID_SIZE);
*/		generate_ints<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,piaAgentBits);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSugar,30.0f);
		generate_floats<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,pfaSpice,30.0f);
		generate_shorts<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(devAgentStates,psaAge,100);
	}

	// initial sugar holdings
	float* pfaInitialSugar;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSugar,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSugar,pfaSugar,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// initial spice holdings
	float* pfaInitialSpice;
	CUDA_CALL(hipMalloc((void**)&pfaInitialSpice,MAX_AGENTS*sizeof(float)));
	CUDA_CALL(hipMemcpy(pfaInitialSpice,pfaSpice,INIT_AGENTS*sizeof(float),hipMemcpyDeviceToDevice));

	// create grid on device
	// sugar in square
	int* pigGridBits;
	CUDA_CALL(hipMalloc((void**)&pigGridBits,GRID_SIZE*GRID_SIZE*sizeof(int)));

	initialize_gridbits<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,pigGridBits);

	// current residents in square - initialized to -1's, aka empty
	int* pigResidents;
	CUDA_CALL(hipMalloc((void**)&pigResidents,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));
	CUDA_CALL(hipMemset(pigResidents,0xFFFF,GRID_SIZE*GRID_SIZE*MAX_OCCUPANCY*sizeof(int)));

	// the agent queue
	int* piaQueueA;
	CUDA_CALL(hipMalloc((void**)&piaQueueA,MAX_AGENTS*sizeof(int)));

	// the deferred queue
	int* piaQueueB;
	CUDA_CALL(hipMalloc((void**)&piaQueueB,MAX_AGENTS*sizeof(int)));

	// the deferred queue size
	int* piDeferredQueueSize;
	CUDA_CALL(hipMalloc((void**)&piDeferredQueueSize,sizeof(int)));

	// the successful locks counter
	int* piLockSuccesses;
	CUDA_CALL(hipMalloc((void**)&piLockSuccesses,sizeof(int)));
	
	// the (dynamic) population counter
	int* piPopulation;
	CUDA_CALL(hipMalloc((void**)&piPopulation,sizeof(int)));
	int* pihPopulation = (int*) malloc(sizeof(int)); 
	pihPopulation[0] = INIT_AGENTS;
	CUDA_CALL(hipMemcpy(piPopulation,pihPopulation,sizeof(int),hipMemcpyHostToDevice));
		
	// position in x
	short* psaX;
	CUDA_CALL(hipMalloc((void**)&psaX,MAX_AGENTS*sizeof(short)));
	short* psahTemp = (short*) malloc(MAX_AGENTS*sizeof(short));
	for (int i = 0; i < INIT_AGENTS; i++) {
		psahTemp[i] = Random()*(GRID_SIZE-1);
	}
	CUDA_CALL(hipMemcpy(psaX,psahTemp,INIT_AGENTS*sizeof(short),hipMemcpyHostToDevice));

	//position in y
	short* psaY;
	CUDA_CALL(hipMalloc((void**)&psaY,MAX_AGENTS*sizeof(short)));
	// fill iTemp arrays with random numbers and copy to device
	for (int i = 0; i < INIT_AGENTS; i++) {
		psahTemp[i] = Random()*(GRID_SIZE-1);
	}
	CUDA_CALL(hipMemcpy(psaY,psahTemp,INIT_AGENTS*sizeof(short),hipMemcpyHostToDevice));

	free(psahTemp);

	hipDeviceSynchronize();

	// timing
	hipEvent_t start;
	hipEvent_t end;
	float elapsed_time;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start,0);

	// count occupancy and store residents

	int status = exercise_locks(COUNT,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,psaAge,pigGridBits,
		pigResidents,piaQueueA,piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);
	
	printf ("Counting %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// time movement
	hipEventRecord(start,0);

	// do movement
	status = exercise_locks(MOVE,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,psaAge,pigGridBits, 
		pigResidents,piaQueueA,piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);
	hipDeviceSynchronize();

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);

	printf ("Moving %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// time harvest, meal and aging
	hipEventRecord(start,0);

	harvest<<<GRID_SIZE,GRID_SIZE>>>(devGridStates,psaX,pfaSugar,pfaSpice,pigGridBits,pigResidents);

	eat<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,piaAgentBits,pfaSugar,pfaSpice);

	age<<<hNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaAge);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);

	printf ("Harvesting %d squares and feeding and aging %d agents takes %f milliseconds\n",(int) GRID_SIZE*GRID_SIZE, (int) pihPopulation[0], (float) elapsed_time);

	// time dying
	hipEventRecord(start,0);

	status = exercise_locks(DIE,psaX,psaY,piaAgentBits,pfaSugar,pfaSpice,psaAge,pigGridBits, 
		pigResidents,piaQueueA,piPopulation,pihPopulation,piaQueueB,piDeferredQueueSize,piLockSuccesses);

	//   end timing
	hipDeviceSynchronize();
	hipEventSynchronize(end);
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed_time, start, end);

	CUDA_CALL(hipMemcpy(pihPopulation,piPopulation,sizeof(int),hipMemcpyDeviceToHost));
	printf ("Registering deaths among %d agents takes %f milliseconds\n",(int) pihPopulation[0], (float) elapsed_time);

	// Cleanup 
	CUDA_CALL(hipFree(psaX));
	CUDA_CALL(hipFree(psaY));
	CUDA_CALL(hipFree(piaAgentBits));
	CUDA_CALL(hipFree(pfaSugar));
	CUDA_CALL(hipFree(pfaSpice));
	CUDA_CALL(hipFree(psaAge));
	CUDA_CALL(hipFree(pfaInitialSugar));
	CUDA_CALL(hipFree(pfaInitialSpice));
	CUDA_CALL(hipFree(pigGridBits));
	CUDA_CALL(hipFree(pigResidents));
	CUDA_CALL(hipFree(piaQueueA));
	CUDA_CALL(hipFree(piaQueueB));
	CUDA_CALL(hipFree(piDeferredQueueSize));
	CUDA_CALL(hipFree(piLockSuccesses));
	CUDA_CALL(hipFree(devAgentStates));
	CUDA_CALL(hipFree(devGridStates));

	return EXIT_SUCCESS;
}
