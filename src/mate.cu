#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "move.h"
#include "mate.h"

const int agentLockMask = 			0x80000000;
const int isFemaleMask = 			0x40000000;
const int visionMask = 				0x30000000;
const int metSugarMask =			0x0C000000;
const int metSpiceMask =			0x03000000;
const int ageMask = 				0x00FE0000;
const int startFertilityAgeMask = 	0x00018000;
const int endFertilityAgeMask = 	0x00007800;

__noinline__ __device__ bool is_fertile_masked(int iAgentID, int* pbaAgentBits, short* psaX)
{
	int age = ((pbaAgentBits[iAgentID])&ageMask)>>17;
	int startFertilityAge = ((pbaAgentBits[iAgentID])&startFertilityAgeMask)>>15;
	int endFertilityAge = ((pbaAgentBits[iAgentID])&endFertilityAgeMask)>>11;
	bool ydResult = (psaX[iAgentID] > -1) &&
			(age > startFertilityAge + 12) &&
			(age < 2*endFertilityAge + 40);
//	if (ydResult) printf("agent %d is fertile\n",iAgentID);
	return ydResult;
}

__noinline__ __device__ bool is_acceptable_mate_masked(int iMateID, int* pbaAgentBits, short* psaX)
{
	bool acceptable = false;

	// make sure this is not an "empty" placeholder
	acceptable = (iMateID > -1 &&
		// make sure he's male
		((pbaAgentBits[iMateID])&isFemaleMask)>>30 == 0 &&
		// and alive
		psaX[iMateID] > -1 &&
		// and fertile
		is_fertile_masked(iMateID,pbaAgentBits,psaX));
	return acceptable;
}
__noinline__ __device__ bool lock_potential_mate_masked(int iMateID, int* pbaAgentBits)
{
	bool lockSuccess = false;

	// if he's unlocked...
	if ((pbaAgentBits[iMateID])&agentLockMask == 0) {
		// make a copy, but indicating locked
		int iTemp = pbaAgentBits[iMateID];
		int iTempLocked = iTemp|agentLockMask;
		// now lock him if possible
		int iLocked = atomicCAS(&(pbaAgentBits[iMateID]),iTemp,iTempLocked);

		// test if the lock worked
		if (iLocked == iTemp) {
			lockSuccess = true;
		}
	}
	return lockSuccess;
}
/*
__noinline__ __device__ bool is_fertile(int iAgentID, AgentBitWise* abwBits, short* psaX)
{
	bool ydResult = (psaX[iAgentID] > -1) &&
			(abwBits->asBits.age > (abwBits->asBits.startFertilityAge + 12)) &&
			(abwBits->asBits.age < (2*abwBits->asBits.endFertilityAge + 40));
	return ydResult;
}

__noinline__ __device__ bool is_acceptable_mate(int iMateID, AgentBitWise* abwBits, short* psaX)
{
	bool acceptable = false;

	// make sure this is not an "empty" placeholder
	acceptable = (iMateID > -1 &&
		// make sure he's male
		abwBits->asBits.isFemale == 0 &&
		// and alive
		psaX[iMateID] > -1 &&
		// and fertile
		is_fertile(iMateID,abwBits,psaX));
	return acceptable;
}

__noinline__ __device__ bool lock_potential_mate(int iMateID, short* psaX, int* pbaBits, AgentBitWise* abwBits)
{
	bool lockSuccess = false;

	// if he's unlocked...
	if (abwBits->asBits.isLocked == 0) {
		// make a copy, but indicating locked
		AgentBitWise abwBitsCopy;
		abwBitsCopy.asInt = abwBits->asInt;
		abwBitsCopy.asBits.isLocked = 1;

		// now lock him if possible
		int iLocked = atomicCAS(&(pbaBits[iMateID]),abwBits->asInt,abwBitsCopy.asInt);

		// test if the lock worked
		if (iLocked == abwBits->asInt) {
			lockSuccess = true;
			abwBits->asInt = abwBitsCopy.asInt;
		}
	}
	return lockSuccess;
}
/*
__global__ void mate_once(short* psaX, short* psaY, int* pbaAgentBits, unsigned int* piaRandoms,
		float* pfaSugar, float* pfaSpice, float* pfaInitialSugar, float* pfaInitialSpice,
		int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, int* piPopulation, int* piaDeferredQueue,
		int* piDeferredQueueSize, int* piLockSuccesses)
{
	int iAgentID;
	int iMateID;
	int iAddy;
	int iAddyTry;
	GridBitWise gbwBits;
	GridBitWise gbwBitsTry;
	bool mated = false;
	bool isGridLocked = false;
	bool isMateLocked = false;
	short sOccTry;
	AgentBitWise abwAgentBits;
	AgentBitWise abwMateBits;
	BitUnpacker buRandoms;
	float fTemp = 0;
	int iTemp = 0;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		iAgentID = piaActiveQueue[iOffset];
		buRandoms.asUInt = piaRandoms[iAgentID];
		abwAgentBits.asInt = pbaAgentBits[iAgentID];

		// live, fertile, solvent female agents only
		if (is_fertile(iAgentID,&abwAgentBits,psaX) && abwAgentBits.asBits.isFemale == 1 &&
				(pfaSugar[iAgentID] > pfaInitialSugar[iAgentID]) && (pfaSpice[iAgentID] > pfaInitialSpice[iAgentID])) {
			iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
			// need to have room on the grid for the kid
			isGridLocked = lock(iAddy,&gbwBits,pigGridBits);
			if (isGridLocked) {
				if (gbwBits.asBits.occupancy < MAX_OCCUPANCY) {
					printf("%d\n",iOffset);
					// get nearest neighbors
					for (short i = -1; i<= 1; i++) {
						for (short j = -1; j <= 1; j++) {
							iAddyTry = (psaX[iAgentID]+i)*GRID_SIZE+psaY[iAgentID]+j;
							gbwBitsTry.asInt = pigGridBits[iAddyTry];
							for (sOccTry = 0; sOccTry < gbwBitsTry.asBits.occupancy; sOccTry++) {
								// note that "mated" terminates the search for mates
								if (!mated) {
									// get the potential mate's id
									iMateID = pigResidents[iAddyTry*MAX_OCCUPANCY+sOccTry];

									// vet his internal properties
									abwMateBits.asInt = pbaAgentBits[iMateID];
/*									if (is_acceptable_mate(iMateID,&abwMateBits,psaX)) {
										// if acceptable, try to lock him
										isMateLocked = lock_potential_mate(iMateID,psaX,pbaAgentBits,&abwMateBits);

										// to get to this point isGridLocked must be true, so this is a logical AND
										if (isMateLocked) {
											// now he's locked, check his solvency
											if	((pfaSugar[iMateID] > pfaInitialSugar[iMateID]) && (pfaSpice[iMateID] > pfaInitialSpice[iMateID])) {
												// ok, he's a keeper
												// mark "mated" only when mate is fully vetted, that way if lock fails, other potential mates are still screened
												mated = true;
												// note that both locks succeeded
												iTemp = atomicAdd(piLockSuccesses,1);

												// get baby's id
												int iChildID = atomicAdd(&(piPopulation[0]),1);

												// insert baby in the grid
												insert_resident(&(gbwBits.asInt),iAddy,pigResidents,psaX,psaY,psaX[iAgentID],psaY[iAgentID],iChildID);

												// set internal properties of baby
												AgentBitWise abwBaby;
												abwBaby.asBits.age = 0;
												abwBaby.asBits.pad = 0;
												abwBaby.asBits.isLocked = 0;

												// baby's sex is random
												abwBaby.asBits.isFemale = buRandoms.asBits.b16;

												// baby's vision and metabolism are inherited from one parent or the other, at random
												if (buRandoms.asBits.b2 == 0) {
													abwBaby.asBits.vision = abwAgentBits.asBits.vision;
												} else {
													abwBaby.asBits.vision = abwMateBits.asBits.vision;
												}
												if (buRandoms.asBits.b3 == 0) {
													abwBaby.asBits.metSugar = abwAgentBits.asBits.metSugar;
												} else {
													abwBaby.asBits.metSugar = abwMateBits.asBits.metSugar;
												}
												if (buRandoms.asBits.b4 == 0) {
													abwBaby.asBits.metSpice = abwAgentBits.asBits.metSpice;
												} else {
													abwBaby.asBits.metSpice = abwMateBits.asBits.metSpice;
												}

												// baby's fertility ages and life expectancy are random (for now)
												abwBaby.asBits.startFertilityAge = buRandoms.asBits.b5+2*buRandoms.asBits.b6;
												abwBaby.asBits.endFertilityAge = buRandoms.asBits.b7+2*buRandoms.asBits.b8+
														4*buRandoms.asBits.b9+8*buRandoms.asBits.b10;
												abwBaby.asBits.deathAge = buRandoms.asBits.b11+2*buRandoms.asBits.b12+
														4*buRandoms.asBits.b13+8*buRandoms.asBits.b14+16*buRandoms.asBits.b15;
												iTemp = atomicExch(&(pbaAgentBits[iChildID]),abwBaby.asInt);

												// baby gets all assets each parent has, up to 5 units of each
												fTemp = min(5.0f,pfaSugar[iAgentID]);
												pfaSugar[iChildID] = fTemp;
												pfaSugar[iAgentID] -= fTemp;
												fTemp = min(5.0f,pfaSugar[iMateID]);
												pfaSugar[iChildID] += fTemp;
												pfaSugar[iMateID] -= fTemp;
												fTemp = min(5.0f,pfaSpice[iAgentID]);
												pfaSpice[iChildID] = fTemp;
												pfaSpice[iAgentID] -= fTemp;
												fTemp = min(5.0f,pfaSpice[iMateID]);
												pfaSpice[iChildID] += fTemp;
												pfaSpice[iMateID] -= fTemp;
												pfaInitialSugar[iChildID] = pfaSugar[iChildID];
												pfaInitialSpice[iChildID] = pfaSpice[iChildID];
												// TODO: give both parents memory of child's id for future inheritance
											}
											// unlock mate
											iTemp = atomicExch(&(pbaAgentBits[iMateID]),abwMateBits.asInt);
										}
									}
					*/		/*	}
							}
						}
					}
				} else {
					// if square is already full, indicate an error
					printf("over occupancy %d to x:%d y:%d\n",gbwBits.asBits.occupancy,psaX[iAgentID],psaY[iAgentID]);
				}
				// unlock square and update global occupancy values
				gbwBits.asBits.isLocked = 0;
				iTemp = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
			}
			// if either lock failed, add the agent to the deferred queue
			if (!isGridLocked || !isMateLocked) {
				iTemp = atomicAdd(piDeferredQueueSize,1);
				piaDeferredQueue[iTemp]=iAgentID;
			}
		}
	}
	return;
}
*/
__global__ void mate_masked(short* psaX, short* psaY, int* pbaAgentBits, unsigned int* piaRandoms,
		float* pfaSugar, float* pfaSpice, float* pfaInitialSugar, float* pfaInitialSpice,
		int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, int* piPopulation, int* piaDeferredQueue,
		int* piDeferredQueueSize, int* piLockSuccesses)
{
	int iAgentID;
	int iMateID;
	int iAddy;
	int iAddyTry;
	GridBitWise gbwBits;
	GridBitWise gbwBitsTry;
	bool mated = false;
	bool isGridLocked = false;
	bool isMateLocked = false;
	short sOccTry;
	float fTemp = 0;
	int iTemp = 0;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		iAgentID = piaActiveQueue[iOffset];

		// live, fertile, solvent female agents only
		printf("fertile %d bits %x\n",is_fertile_masked(iAgentID,pbaAgentBits,psaX),(pbaAgentBits[iAgentID])); //((pbaAgentBits[iAgentID])&isFemaleMask)>>30); // == 1 &&
//				(pfaSugar[iAgentID] > pfaInitialSugar[iAgentID]) && (pfaSpice[iAgentID] > pfaInitialSpice[iAgentID])) {
		/*	iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
			// need to have room on the grid for the kid
			isGridLocked = lock(iAddy,&gbwBits,pigGridBits);
			if (isGridLocked) {
				if (gbwBits.asBits.occupancy < MAX_OCCUPANCY) {
					// get nearest neighbors
					for (short i = -1; i<= 1; i++) {
						for (short j = -1; j <= 1; j++) {
							iAddyTry = (psaX[iAgentID]+i)*GRID_SIZE+psaY[iAgentID]+j;
							gbwBitsTry.asInt = pigGridBits[iAddyTry];
							for (sOccTry = 0; sOccTry < gbwBitsTry.asBits.occupancy; sOccTry++) {
								// note that "mated" terminates the search for mates
								if (!mated) {
									// get the potential mate's id
									iMateID = pigResidents[iAddyTry*MAX_OCCUPANCY+sOccTry];
									printf("%d\n",iOffset);
									// vet his internal properties
									if (is_acceptable_mate_masked(iMateID,pbaAgentBits,psaX)) {
										// if acceptable, try to lock him
										isMateLocked = lock_potential_mate_masked(iMateID,pbaAgentBits);

										// to get to this point isGridLocked must be true, so this is a logical AND
										if (isMateLocked) {
											// now he's locked, check his solvency
											if	((pfaSugar[iMateID] > pfaInitialSugar[iMateID]) && (pfaSpice[iMateID] > pfaInitialSpice[iMateID])) {
												// ok, he's a keeper
												// mark "mated" only when mate is fully vetted, that way if lock fails, other potential mates are still screened
												mated = true;
												// note that both locks succeeded
												iTemp = atomicAdd(piLockSuccesses,1);

												// get baby's id
												int iChildID = atomicAdd(&(piPopulation[0]),1);

												// insert baby in the grid
												insert_resident(&(gbwBits.asInt),iAddy,pigResidents,psaX,psaY,psaX[iAgentID],psaY[iAgentID],iChildID);

												// set internal properties of baby
/*												AgentBitWise abwBaby;
												abwBaby.asBits.age = 0;
												abwBaby.asBits.pad = 0;
												abwBaby.asBits.isLocked = 0;

												// baby's sex is random
												abwBaby.asBits.isFemale = buRandoms.asBits.b16;

												// baby's vision and metabolism are inherited from one parent or the other, at random
												if (buRandoms.asBits.b2 == 0) {
													abwBaby.asBits.vision = abwAgentBits.asBits.vision;
												} else {
													abwBaby.asBits.vision = abwMateBits.asBits.vision;
												}
												if (buRandoms.asBits.b3 == 0) {
													abwBaby.asBits.metSugar = abwAgentBits.asBits.metSugar;
												} else {
													abwBaby.asBits.metSugar = abwMateBits.asBits.metSugar;
												}
												if (buRandoms.asBits.b4 == 0) {
													abwBaby.asBits.metSpice = abwAgentBits.asBits.metSpice;
												} else {
													abwBaby.asBits.metSpice = abwMateBits.asBits.metSpice;
												}

												// baby's fertility ages and life expectancy are random (for now)
												abwBaby.asBits.startFertilityAge = buRandoms.asBits.b5+2*buRandoms.asBits.b6;
												abwBaby.asBits.endFertilityAge = buRandoms.asBits.b7+2*buRandoms.asBits.b8+
														4*buRandoms.asBits.b9+8*buRandoms.asBits.b10;
												abwBaby.asBits.deathAge = buRandoms.asBits.b11+2*buRandoms.asBits.b12+
														4*buRandoms.asBits.b13+8*buRandoms.asBits.b14+16*buRandoms.asBits.b15;
												iTemp = atomicExch(&(pbaAgentBits[iChildID]),abwBaby.asInt);

												// baby gets all assets each parent has, up to 5 units of each
												fTemp = min(5.0f,pfaSugar[iAgentID]);
												pfaSugar[iChildID] = fTemp;
												pfaSugar[iAgentID] -= fTemp;
												fTemp = min(5.0f,pfaSugar[iMateID]);
												pfaSugar[iChildID] += fTemp;
												pfaSugar[iMateID] -= fTemp;
												fTemp = min(5.0f,pfaSpice[iAgentID]);
												pfaSpice[iChildID] = fTemp;
												pfaSpice[iAgentID] -= fTemp;
												fTemp = min(5.0f,pfaSpice[iMateID]);
												pfaSpice[iChildID] += fTemp;
												pfaSpice[iMateID] -= fTemp;
												pfaInitialSugar[iChildID] = pfaSugar[iChildID];
												pfaInitialSpice[iChildID] = pfaSpice[iChildID];
												// TODO: give both parents memory of child's id for future inheritance
											}
											// unlock mate
						//					iTemp = atomicExch(&(pbaAgentBits[iMateID]),abwMateBits.asInt);
										}
									}
								}
							}
						}
					}
				} else {
					// if square is already full, indicate an error
					printf("over occupancy %d to x:%d y:%d\n",gbwBits.asBits.occupancy,psaX[iAgentID],psaY[iAgentID]);
				}
				// unlock square and update global occupancy values
				gbwBits.asBits.isLocked = 0;
				iTemp = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
			}
			// if either lock failed, add the agent to the deferred queue
			if (!isGridLocked || !isMateLocked) {
				iTemp = atomicAdd(piDeferredQueueSize,1);
				piaDeferredQueue[iTemp]=iAgentID;
			}
		}
*/	}
	return;
}
