#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "symbolic_constants.h"
#include "bitwisetype.h"
#include "mate.h"

__device__ bool isFertile(int iAgentID, BitWiseType* pbaBits, short* psaAge)
{
	bool ydResult = (psaX[iAgentID] > -1) &&
			(psaAge[iAgentID] > ((&pbaBits[iAgentID])->startFertilityAge + 12)) &&
			(psaAge[iAgentID] < (2*((&pbaBits[iAgentID])->endFertilityAge) + 40));
	return ydResult;
}
__global__ void mate(short* psaX, short* psaY, short* psgOccupancy, int* pigResidents, int* pigLocks, 
		int* piaActiveQueue, const int ciActiveQueueSize, int* piaDeferredQueue, int* piDeferredQueueSize, 
		int* piLockSuccesses)
{
	int iAgentID;
	int iMateID;
	int iAddy;
	int iTemp;
	short sOldOcc;
	bool pregnant = false;
	bool lockFailed = false;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		iAgentID = piaActiveQueue[iOffset];

		// live, fertile, solvent female agents only
		if (((&pbaBits[iAgentID])->isFemale) == 1) {
			if (isFertile(iAgentID)) {
				if	((pfaSugar[iAgentID] > pfaInitialSugar[iAgentID]) &&
						(pfaSpice[iAgentID] > pfaInitialSpice[iAgentID])) {
					for (short sXTry = psaX[iAgentID] - 1; sXTry < psaX[iAgentID] + 2; sXTry++) {
						for (short sYTry = psaY[iAgentID] - 1; sYTry < psaY[iAgentID] + 2; sYTry++) {
							short sOccTry = 0;
							while (sOccTry < MAX_OCCUPANCY && !pregnant) {
								// get the potential mate's id
								int iMateID = pigResidents[(sXTry*GRID_SIZE+sYTry)*MAX_OCCUPANCY+sOccTry];
								// make sure this is not an "empty" placeholder
								if (iMateID > -1) {
									// make sure he's male, alive and fertile
									if ((&pbaBits[iAgentID])->isFemale == 0 && isFertile(iMateID)) {
										// if he's unlocked...
										short sHisAge = psaAge[iMateID];
										if (sHisAge > 0) {
											// lock him if possible by changing his age to negative
											int iTemp = atomicCAS(&(psaAge[iMateID]),sHisAge,-sHisAge);
											if (iTemp == sHisAge) {
												// now he's locked, check his solvency
												if	((pfaSugar[iMateID] > pfaInitialSugar[iMateID]) &&
														(pfaSpice[iMateID] > pfaInitialSpice[iMateID])) {
													// ok he's a keeper, make a baby...
													pregnant = true;
												} else {
													// unlock him
													iTemp = atomicExch(&(psaAge[iMateID]),abs(sHisAge));
												}
											} else {
												lockFailed = true;
											}
										}
									}
								}
							}
						}
					}
					// TODO: if lock failed anywhere, defer this agent
				}
			}
		}
	}



				// increment square occupancy and counter of successful locks
				sOldOcc = psgOccupancy[iAddy]++;
				iTemp = atomicAdd(piLockSuccesses,1);

				// check for overflow
				if (sOldOcc < MAX_OCCUPANCY) {

					// insert the resident at the next position in the pigResidents list
					pigResidents[iAddy*MAX_OCCUPANCY+sOldOcc] = iAgentID;

				} else {

					// indicate an occupancy overflow
					printf ("overflow at x:%d y:%d \n",psaX[iAgentID], psaY[iAgentID]);
					psgOccupancy[iAddy] = MAX_OCCUPANCY+1;
				}


			}
			else {

				// otherwise, add the agent to the deferred queue
				iTemp = atomicAdd(piDeferredQueueSize,1);
				piaDeferredQueue[iTemp]=iAgentID;
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
__global__ void count_occupancy_fs(short* psaX, short* psaY, short* psgOccupancy, int* pigResidents,
		int* piaActiveQueue, const int ciActiveQueueSize)
{
	int iAgentID;
	int iAddy;
	short sOldOcc;

	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get agent id
			iAgentID = piaActiveQueue[iOffset];

			// work with live agents only
			if (psaX[iAgentID] > -1) {

				// current agent's address in the grid
				iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];

				// no locks necessary, so increment square occupancy
				sOldOcc = psgOccupancy[iAddy]++;

				// check for overflow
				if (sOldOcc < MAX_OCCUPANCY) {

					// insert the resident at the next position in the pigResidents list
					pigResidents[iAddy*MAX_OCCUPANCY+sOldOcc] = iAgentID;

				} else {

					// indicate an occupancy overflow
					printf ("overflow at x:%d y:%d \n",psaX[iAgentID], psaY[iAgentID]);
					psgOccupancy[iAddy] = MAX_OCCUPANCY+1;
				}

			}
		}
	}
	return;
}

int count(short* psaX, short* psaY, short* psgOccupancy, int* pigResidents, int* pigLocks, 
		int* piaQueueA, const int iQueueSize, int* piaQueueB, int* piDeferredQueueSize, int* piLockSuccesses)
{
	int status = EXIT_SUCCESS;


	// fill the agent queue with increasing (later random) id's
	int* piahTemp = (int*) malloc(iQueueSize*sizeof(int));
	for (int i = 0; i < iQueueSize; i++) {
		//		piahTemp[i] = rand() % iQueueSize;
		piahTemp[i] = i;
	}
	CUDA_CALL(hipMemcpy(piaQueueA,piahTemp,iQueueSize*sizeof(int),hipMemcpyHostToDevice));

	// blank the deferred queue with all bits=1
	CUDA_CALL(hipMemset(piaQueueB,0xFF,iQueueSize*sizeof(int)));

	// zero the deferred queue size
	CUDA_CALL(hipMemset(piDeferredQueueSize,0,sizeof(int)));

	// zero the successful locks counter
	CUDA_CALL(hipMemset(piLockSuccesses,0,sizeof(int)));

	// count agents at each grid location
	int hiNumBlocks = (iQueueSize+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
	count_occupancy<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,psgOccupancy,pigResidents,pigLocks,
			piaQueueA,iQueueSize,piaQueueB,piDeferredQueueSize,piLockSuccesses);
	hipDeviceSynchronize();

	// check if any agents had to be deferred
	int* pihDeferredQueueSize = (int*) malloc(sizeof(int));
	CUDA_CALL(hipMemcpy(pihDeferredQueueSize,piDeferredQueueSize,sizeof(int),hipMemcpyDeviceToHost));
	printf ("primary deferrals:%d \n",pihDeferredQueueSize[0]);
	int* pihLockSuccesses = (int*) malloc(sizeof(int));
	CUDA_CALL(hipMemcpy(pihLockSuccesses,piLockSuccesses,sizeof(int),hipMemcpyDeviceToHost));
	printf ("successful locks:%d \n",pihLockSuccesses[0]);

	// handle the deferred queue until it is empty
	int ihActiveQueueSize = iQueueSize;
	bool hQueue = true;
	while (pihDeferredQueueSize[0] > 10 && pihDeferredQueueSize[0] < ihActiveQueueSize) {
		ihActiveQueueSize = pihDeferredQueueSize[0];
		hiNumBlocks = (ihActiveQueueSize+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
		CUDA_CALL(hipMemset(piDeferredQueueSize,0,sizeof(int)));
		CUDA_CALL(hipMemset(piLockSuccesses,0,sizeof(int)));
		if (hQueue) {
			CUDA_CALL(hipMemset(piaQueueA,0xFF,iQueueSize*sizeof(int)));
			count_occupancy<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,psgOccupancy,pigResidents,pigLocks,
					piaQueueB,ihActiveQueueSize,piaQueueA,piDeferredQueueSize,piLockSuccesses);
		}
		else {
			CUDA_CALL(hipMemset(piaQueueB,0xFF,iQueueSize*sizeof(int)));
			count_occupancy<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,psgOccupancy,pigResidents,pigLocks,
					piaQueueA,ihActiveQueueSize,piaQueueB,piDeferredQueueSize,piLockSuccesses);
		}
		hipDeviceSynchronize();
		hQueue = !hQueue;
		CUDA_CALL(hipMemcpy(pihDeferredQueueSize,piDeferredQueueSize,sizeof(int),hipMemcpyDeviceToHost));
		printf ("secondary deferrals:%d \n",pihDeferredQueueSize[0]);
	}

	// for persistent lock failures, use the failsafe version
	if (pihDeferredQueueSize[0] <= 10 || pihDeferredQueueSize[0] >= ihActiveQueueSize) {
		ihActiveQueueSize = pihDeferredQueueSize[0];
		if (hQueue) {
			count_occupancy_fs<<<1,1>>>(psaX,psaY,psgOccupancy,pigResidents,piaQueueB,ihActiveQueueSize);

		} else {
			count_occupancy_fs<<<1,1>>>(psaX,psaY,psgOccupancy,pigResidents,piaQueueA,ihActiveQueueSize);
		}
		hipDeviceSynchronize();
	}

	// check for overflows
	short* psghOccupancy = (short*) malloc(GRID_SIZE*GRID_SIZE*sizeof(short));
	CUDA_CALL(hipMemcpy(psghOccupancy,psgOccupancy,GRID_SIZE*GRID_SIZE*sizeof(short),hipMemcpyDeviceToHost));
	for (int k = 0; k < GRID_SIZE*GRID_SIZE; k++) {
		if (psghOccupancy[k] > MAX_OCCUPANCY) {
			printf ("Occupancy overflow at square %d\n",k);
			status = OCCUPANCY_OVERFLOW;
		}
	}

	// cleanup
	free(pihLockSuccesses);
	free(pihDeferredQueueSize);
	free(psghOccupancy);
	free(piahTemp);

	return status;
}
