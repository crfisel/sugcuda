#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "move.h"

// this kernel has one thread per agent, each traversing the local neighborhood prescribed by its vision
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void best_move_by_traversal(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, 
	float* pfaSpice, int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, 
	int* piaDeferredQueue, int* piDeferredQueueSize, int* piLockSuccesses)
{
	GridBitWise gbwBits;
	int iFlag = 0;
	bool lockFailed = false;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		int iAgentID = piaActiveQueue[iOffset];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

#include "traversal_routine.cu"

			// if a move is warranted, lock old and new address - if either fails, defer
			if (sXStore != sXCenter || sYStore != sYCenter) {

				// agent's current address in the grid
				int iOldAddy = sXCenter*GRID_SIZE+sYCenter;
				// unpack grid bits
				gbwBits.asInt = pigGridBits[iOldAddy];
			
				// test if old square is locked
				if (gbwBits.asBits.isLocked != 0) {
					// if so, lock failed
					lockFailed = true;
				} else {
					// if not, make a copy, but indicating locked
					GridBitWise gbwBitsCopy = gbwBits;
					gbwBitsCopy.asBits.isLocked = 1;

					// now lock the current address if possible
					int iLockedOld = atomicCAS(&(pigGridBits[iOldAddy]),gbwBits.asInt,gbwBitsCopy.asInt);
					// test if the lock failed
					if (iLockedOld != gbwBits.asInt) {
						lockFailed = true;
					} else {
						// at this point, old square is locked and a valid copy of its bits are in gbwBitsCopy (because locked)
						// agent's new address in the grid
						int iNewAddy = sXStore*GRID_SIZE+sYStore;
						// unpack grid bits
						GridBitWise gbwNewBits;
						gbwNewBits.asInt = pigGridBits[iNewAddy];
			
						// test if new square is locked
						if (gbwNewBits.asBits.isLocked != 0) {
							// if so, lock failed
							lockFailed = true;
							// unlock old square by replacing the old (unlocked) bits
							iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);

						} else {
							// if not, make a copy, but indicating locked
							GridBitWise gbwNewBitsCopy = gbwNewBits;
							gbwNewBitsCopy.asBits.isLocked = 1;

							// now lock the new address if possible
							int iLockedNew = atomicCAS(&(pigGridBits[iNewAddy]),gbwNewBits.asInt,gbwNewBitsCopy.asInt);

							// test if the lock failed
							if (iLockedNew != gbwNewBits.asInt) {
								lockFailed = true;
								// unlock old square by replacing the old (unlocked) bits
								iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
							} else {
								// at this point the squares are locked and valid copies are in the "copy" variables
								iFlag = atomicAdd(piLockSuccesses,1);

								// before inserting new resident, check for nonzero old occupancy (negatives forbidden by unsigned short declaration)
								// and make sure new address is not already full 
								if (gbwBitsCopy.asBits.occupancy <= 0 || 
										gbwNewBitsCopy.asBits.occupancy >= MAX_OCCUPANCY) {
									
									// unlock with no changes
									iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBits.asInt);
									iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
									
									// indicate an error
									printf("over occ %d at x:%d y:%d or under occ %d at x:%d y:%d agent %d\n",
										gbwNewBitsCopy.asBits.occupancy,sXStore,sYStore,gbwBitsCopy.asBits.occupancy,sXCenter,sYCenter,iAgentID);
								} else {
									// find match starting at end of list
									short k = --gbwBitsCopy.asBits.occupancy;

									// remove current id - if not at the end, replace it with the one from the end and store -1 at end
									if (pigResidents[iOldAddy*MAX_OCCUPANCY+k] == iAgentID) {
										pigResidents[iOldAddy*MAX_OCCUPANCY+k] = -1;
									} else {
										while (pigResidents[iOldAddy*MAX_OCCUPANCY+k] != iAgentID && k >= 0) {k--;}
										if (k != gbwBitsCopy.asBits.occupancy) {
											pigResidents[iOldAddy*MAX_OCCUPANCY+k] = pigResidents[iOldAddy*MAX_OCCUPANCY+gbwBitsCopy.asBits.occupancy];
											pigResidents[iOldAddy*MAX_OCCUPANCY+gbwBitsCopy.asBits.occupancy] = -1;
										}
									}

									// make sure we are replacing an "empty" placemarker
									if (pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBitsCopy.asBits.occupancy] == -1) {
										psaX[iAgentID] = sXStore;
										psaY[iAgentID] = sYStore;
										pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBitsCopy.asBits.occupancy] = iAgentID;

										// increment occupancy at new address
										gbwNewBitsCopy.asBits.occupancy++;
									} else {

										//otherwise notify about the error
										printf ("agent replaced %d at x:%d y:%d \n",
										pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBitsCopy.asBits.occupancy],
											sXStore,sYStore);
									}
								}  
								// unlock and update global occupancy values
								gbwNewBitsCopy.asBits.isLocked = 0;
								iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBitsCopy.asInt);
								gbwBitsCopy.asBits.isLocked = 0;
								iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBitsCopy.asInt);
							} 
						}
					}
				}
				// if a move was warranted, but lock failures prevented it, defer
				if (lockFailed) {
					// if either lock failed or either agent was already locked, add the agent to the deferred queue
					iFlag = atomicAdd(piDeferredQueueSize,1);
					piaDeferredQueue[iFlag]=iAgentID;
				}
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void best_move_by_traversal_fs(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, 
	float* pfaSpice, int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize)
{
	GridBitWise gbwBits;
	int iAgentID;

	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get the iAgentID from the active agent queue
			iAgentID = piaActiveQueue[iOffset];

			// if the agent is alive
			if (psaX[iAgentID] > -1) {

#include "traversal_routine.cu"

				if (sXStore != sXCenter || sYStore != sYCenter) {

					// if a move is warranted, go, no need to lock
					// get old and new addresses in the grid
					int iOldAddy = sXCenter*GRID_SIZE+sYCenter;
					int iNewAddy = sXStore*GRID_SIZE+sYStore;

					// unpack grid bits
					gbwBits.asInt = pigGridBits[iOldAddy];
					GridBitWise gbwNewBits;
					gbwNewBits.asInt = pigGridBits[iNewAddy];

					// before inserting new resident, check for nonzero old occupancy (negatives forbidden by unsigned short declaration)
					// and make sure new address is not already full 
					if (gbwBits.asBits.occupancy <= 0 || 
						gbwNewBits.asBits.occupancy >= MAX_OCCUPANCY) {
									
						// indicate an error
						printf("over occ %d at x:%d y:%d or under occ %d at x:%d y:%d agent %d\n",
							gbwNewBits.asBits.occupancy,sXStore,sYStore,gbwBits.asBits.occupancy,sXCenter,sYCenter,iAgentID);

					} else {			
						// find match starting at end of list
						short k = --gbwBits.asBits.occupancy;

						// remove current id - if not at the end, replace it with the one from the end and store -1 at end
						if (pigResidents[iOldAddy*MAX_OCCUPANCY+k] == iAgentID) {
							pigResidents[iOldAddy*MAX_OCCUPANCY+k] = -1;
						} else {
							while (pigResidents[iOldAddy*MAX_OCCUPANCY+k] != iAgentID && k >= 0) {k--;}
							if (k != gbwBits.asBits.occupancy) {
								pigResidents[iOldAddy*MAX_OCCUPANCY+k] = pigResidents[iOldAddy*MAX_OCCUPANCY+gbwBits.asBits.occupancy];
								pigResidents[iOldAddy*MAX_OCCUPANCY+gbwBits.asBits.occupancy] = -1;
							}
						}							

						// make sure we are replacing an "empty" placemarker
						if (pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBits.asBits.occupancy] == -1) {
							psaX[iAgentID] = sXStore;
							psaY[iAgentID] = sYStore;
							pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBits.asBits.occupancy] = iAgentID;

							// increment occupancy at new address
							gbwNewBits.asBits.occupancy++;
						} else {

							//otherwise notify about the error
							printf ("agent replaced %d at x:%d y:%d \n",
							pigResidents[iNewAddy*MAX_OCCUPANCY+gbwNewBits.asBits.occupancy],
								sXStore,sYStore);
						}
					}
					// update global occupancy values
					int iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBits.asInt);
					iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
				}
			}
		}
	}
	return;
}

