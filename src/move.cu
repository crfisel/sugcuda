#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "move.h"

// this kernel has one thread per agent, each traversing the local neighborhood prescribed by its vision
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void best_move_by_traversal(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, 
	float* pfaSpice, int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, 
	int* piaDeferredQueue, int* piDeferredQueueSize, int* piLockSuccesses)
{
	int iFlag = 0;
	bool lockFailed = false;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		int iAgentID = piaActiveQueue[iOffset];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

#include "traversal_routine.cu"

			// if a move is warranted, lock old and new address - if either fails, defer
			if (sXStore != sXCenter || sYStore != sYCenter) {

				// agent's current address in the grid
				int iOldAddy = sXCenter*GRID_SIZE+sYCenter;
				// unpack grid bits
				GridBitWise gbwBits;
				gbwBits.asInt = pigGridBits[iOldAddy];
			
				// test if old square is locked
				if (gbwBits.asBits.isLocked != 0) {
					// if so, lock failed
					lockFailed = true;
				} else {
					// if not, make a copy, but indicating locked
					GridBitWise gbwBitsCopy = gbwBits;
					gbwBitsCopy.asBits.isLocked = 1;

					// now lock the current address if possible
					int iLockedOld = atomicCAS(&(pigGridBits[iOldAddy]),gbwBits.asInt,gbwBitsCopy.asInt);
					// test if the lock failed
					if (iLockedOld != gbwBits.asInt) {
						lockFailed = true;
					} else {
						// at this point, old square is locked and a valid copy of its bits are in gbwBitsCopy (because locked)
						// agent's new address in the grid
						int iNewAddy = sXStore*GRID_SIZE+sYStore;
						// unpack grid bits
						GridBitWise gbwNewBits;
						gbwNewBits.asInt = pigGridBits[iNewAddy];
			
						// test if new square is locked
						if (gbwNewBits.asBits.isLocked != 0) {
							// if so, lock failed
							lockFailed = true;
							// unlock old square by replacing the old (unlocked) bits
							iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);

						} else {
							// if not, make a copy, but indicating locked
							GridBitWise gbwNewBitsCopy = gbwNewBits;
							gbwNewBitsCopy.asBits.isLocked = 1;

							// now lock the new address if possible
							int iLockedNew = atomicCAS(&(pigGridBits[iNewAddy]),gbwNewBits.asInt,gbwNewBitsCopy.asInt);

							// test if the lock failed
							if (iLockedNew != gbwNewBits.asInt) {
								lockFailed = true;
								// unlock old square by replacing the old (unlocked) bits
								iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
							} else {
								// at this point the squares are locked and valid copies are in the "copy" variables
								iFlag = atomicAdd(piLockSuccesses,1);

								// before inserting new resident, check for nonzero old occupancy (negatives forbidden by unsigned short declaration)
								// and make sure new address is not already full 
								if (gbwBitsCopy.asBits.occupancy <= 0 || 
									gbwNewBitsCopy.asBits.occupancy >= MAX_OCCUPANCY) {
									
									// unlock with no changes
									iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBits.asInt);
									iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
									
									// indicate an error
									printf("over occ %d at x:%d y:%d or under occ %d at x:%d y:%d agent %d\n",
										gbwNewBitsCopy.asBits.occupancy,sXStore,sYStore,gbwBitsCopy.asBits.occupancy,sXCenter,sYCenter,iAgentID);
								} else {
									remove_resident(&(gbwBitsCopy.asInt),iOldAddy,pigResidents,iAgentID);
									insert_resident(&(gbwNewBitsCopy.asInt),iNewAddy,pigResidents,psaX,psaY,sXStore,sYStore,iAgentID);
								} 
								// unlock and update global occupancy values
								gbwNewBitsCopy.asBits.isLocked = 0;
								iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBitsCopy.asInt);
								gbwBitsCopy.asBits.isLocked = 0;
								iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBitsCopy.asInt);
							}
						}
					}
				}
				// if a move was warranted, but lock failures prevented it, defer
				if (lockFailed) {
					// if either lock failed or either agent was already locked, add the agent to the deferred queue
					iFlag = atomicAdd(piDeferredQueueSize,1);
					piaDeferredQueue[iFlag]=iAgentID;
				}
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void best_move_by_traversal_fs(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, 
	float* pfaSpice, int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize)
{

	int iAgentID;

	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get the iAgentID from the active agent queue
			iAgentID = piaActiveQueue[iOffset];

			// if the agent is alive
			if (psaX[iAgentID] > -1) {

#include "traversal_routine.cu"

				if (sXStore != sXCenter || sYStore != sYCenter) {

					// if a move is warranted, go, no need to lock
					// get old and new addresses in the grid
					int iOldAddy = sXCenter*GRID_SIZE+sYCenter;
					int iNewAddy = sXStore*GRID_SIZE+sYStore;

					// unpack grid bits
					GridBitWise gbwBits;
					gbwBits.asInt = pigGridBits[iOldAddy];
					GridBitWise gbwNewBits;
					gbwNewBits.asInt = pigGridBits[iNewAddy];

					// before inserting new resident, check for nonzero old occupancy (negatives forbidden by unsigned short declaration)
					// and make sure new address is not already full 
					if (gbwBits.asBits.occupancy <= 0 || 
						gbwNewBits.asBits.occupancy >= MAX_OCCUPANCY) {
									
						// indicate an error
						printf("over occ %d at x:%d y:%d or under occ %d at x:%d y:%d agent %d\n",
							gbwNewBits.asBits.occupancy,sXStore,sYStore,gbwBits.asBits.occupancy,sXCenter,sYCenter,iAgentID);

					} else {			

						remove_resident(&(gbwBits.asInt),iOldAddy,pigResidents,iAgentID);
						insert_resident(&(gbwNewBits.asInt),iNewAddy,pigResidents,psaX,psaY,sXStore,sYStore,iAgentID);

						// update global occupancy values
						int iFlag = atomicExch(&(pigGridBits[iNewAddy]),gbwNewBits.asInt);
						iFlag = atomicExch(&(pigGridBits[iOldAddy]),gbwBits.asInt);
					}
				}
			}
		}
	}
	return;
}

