#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "die.h"

// this kernel has one thread per agent, each traversing the local neighborhood prescribed by its vision
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void register_deaths(short* psaX, short* psaY, int* piaBits, short* psaAge,
		float* pfaSugar, float* pfaSpice, short* psgOccupancy, int* pigResidents,
		int* pigLocks, int* piaActiveQueue, const int ciActiveQueueSize,
		int* piaDeferredQueue, int* piDeferredQueueSize, int* piLockSuccesses)
{
	int iAgentID;
	int iTemp = 0;
	int iFlag = 0;
	int iLocked = 0;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		iAgentID = piaActiveQueue[iOffset];

		// if agent is alive
		if (psaX[iAgentID] > -1) {

			// check for death by old age
			// reinterpret piaBits bitwise for death age
			BitWise bwLocalBits;
			bwLocalBits.asInt = piaBits[iAgentID];
			if ((psaAge[iAgentID] > 64+(bwLocalBits.asBits.deathAge)) ||
					// check for starvation
					(pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {
				//printf("age %d death age %d sugar %f spice %f\n",psaAge[iAgentID],60+2*((&piaBits[iAgentID])->deathAge),pfaSugar[iAgentID],pfaSpice[iAgentID]);
				// lock address to register death - if lock fails, defer

				// current agent's address in the grid
				int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
				iLocked = atomicCAS(&(pigLocks[iAddy]), 0, 1);
				if (iLocked == 0) {
					iFlag = atomicAdd(piLockSuccesses,1);
					//	printf("death %d at %d:%d\n",iAgentID,psaX[iAgentID],psaY[iAgentID]);

					// decrement occupancy at old address
					short sOldOcc = psgOccupancy[iAddy]--;
					if (sOldOcc >= 0) { 

						// find match starting at end of list
						short k = sOldOcc;
						while (pigResidents[iAddy*MAX_OCCUPANCY+k] != iAgentID && k > 0) {k--;} //PROBLEM HERE!!!!

						// remove current id - if it is not at the end, replace it with the one from the end
						if (k != sOldOcc) atomicExch(&(pigResidents[iAddy*MAX_OCCUPANCY+k]), 
								pigResidents[iAddy*MAX_OCCUPANCY+sOldOcc]);
					} else {

						// in case of bugs (i.e. old occupancy was already zero), report problem
						//printf ("underflow at x:%d y:%d \n",psaX[iAgentID],psaY[iAgentID]);
					}
					// mark agent as dead
					psaX[iAgentID] *= -1;
				} else {

					// otherwise, add the agent to the deferred queue
					iTemp = atomicAdd(piDeferredQueueSize,1);
					piaDeferredQueue[iTemp]=iAgentID;
				}

				// unlock
				iFlag = atomicExch(&(pigLocks[iAddy]),0);
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
// NOTE: NUM_AGENTS is an int, GRID_SIZE is a short
__global__ void register_deaths_fs(short* psaX, short* psaY, int* piaBits, short* psaAge,
		float* pfaSugar, float* pfaSpice, short* psgOccupancy, int* pigResidents, int* piaActiveQueue,
		const int ciActiveQueueSize)
{
	int iAgentID;
	int iTemp = 0;

	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get the iAgentID from the active agent queue
			iAgentID = piaActiveQueue[iOffset];

			// if agent is alive
			if (psaX[iAgentID] > -1) {

				// check for death by old age
				// reinterpret piaBits bitwise for death age
				BitWise bwLocalBits;
				bwLocalBits.asInt = piaBits[iAgentID];
				if ((psaAge[iAgentID] > 64+(bwLocalBits.asBits.deathAge)) ||
						// check for starvation
						(pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {
					//printf("age %d sugar %f spice %f\n",psaAge[iAgentID],pfaSugar[iAgentID],pfaSpice[iAgentID]);
					// current agent's address in the grid
					int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
					//	printf("death %d at %d:%d\n",iAgentID,psaX[iAgentID],psaY[iAgentID]);

					// decrement occupancy at old address
					short sOldOcc = psgOccupancy[iAddy]--;
					if (sOldOcc >= 0) { 

						// find match starting at end of list
						short k = sOldOcc;
						while (pigResidents[iAddy*MAX_OCCUPANCY+k] != iAgentID && k > 0) {k--;} //PROBLEM HERE!!!!

						// remove current id - if it is not at the end, replace it with the one from the end
						if (k != sOldOcc) atomicExch(&(pigResidents[iAddy*MAX_OCCUPANCY+k]), 
								pigResidents[iAddy*MAX_OCCUPANCY+sOldOcc]);
					} else {

						// in case of bugs (i.e. old occupancy was already zero), report problem
						printf ("underflow at x:%d y:%d \n",psaX[iAgentID],psaY[iAgentID]);
					}
					// mark agent as dead
					psaX[iAgentID] *= -1;	
				}
			}
		}
	}
	return;
}

int die(short* psaX, short* psaY, int* piaBits, short* psaAge, float* pfaSugar, float* pfaSpice, short* psgOccupancy, 
		int* pigResidents, int* pigLocks, int* piaQueueA, const int iQueueSize, int* piaQueueB, int* piDeferredQueueSize,
		int* piLockSuccesses)
{
	int status = EXIT_SUCCESS;

	// fill the agent queue with increasing (later random) id's
	int* piahTemp = (int*) malloc(iQueueSize*sizeof(int));
	for (int i = 0; i < iQueueSize; i++) {
		//		piahTemp[i] = rand() % iQueueSize;
		piahTemp[i] = i;
	}
	CUDA_CALL(hipMemcpy(piaQueueA,piahTemp,iQueueSize*sizeof(int),hipMemcpyHostToDevice));

	// blank the deferred queue with all bits=1
	CUDA_CALL(hipMemset(piaQueueB,0xFF,iQueueSize*sizeof(int)));

	// zero the deferred queue size
	CUDA_CALL(hipMemset(piDeferredQueueSize,0,sizeof(int)));

	// zero the successful locks counter
	CUDA_CALL(hipMemset(piLockSuccesses,0,sizeof(int)));

	// find best move for agents at the head of their square's resident list
	int hiNumBlocks = (iQueueSize+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
	register_deaths<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,
			psgOccupancy,pigResidents,pigLocks,piaQueueA,iQueueSize,piaQueueB,piDeferredQueueSize,piLockSuccesses);
	hipDeviceSynchronize();

	// check if any agents had to be deferred
	int* pihDeferredQueueSize = (int*) malloc(sizeof(int));
	CUDA_CALL(hipMemcpy(pihDeferredQueueSize,piDeferredQueueSize,sizeof(int),hipMemcpyDeviceToHost));
	printf ("primary deferrals:%d \n",pihDeferredQueueSize[0]);
	int* pihLockSuccesses = (int*) malloc(sizeof(int));
	CUDA_CALL(hipMemcpy(pihLockSuccesses,piLockSuccesses,sizeof(int),hipMemcpyDeviceToHost));
	printf ("successful locks:%d \n",pihLockSuccesses[0]);


	// handle the deferred queue until it is empty
	int ihActiveQueueSize = iQueueSize;
	bool hQueue = true;
	while (pihDeferredQueueSize[0] > 100 && pihDeferredQueueSize[0] < ihActiveQueueSize) {
		ihActiveQueueSize = pihDeferredQueueSize[0];
		hiNumBlocks = (ihActiveQueueSize+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK;
		CUDA_CALL(hipMemset(piDeferredQueueSize,0,sizeof(int)));
		CUDA_CALL(hipMemset(piLockSuccesses,0,sizeof(int)));
		if (hQueue) {
			CUDA_CALL(hipMemset(piaQueueA,0xFF,iQueueSize*sizeof(int)));
			register_deaths<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,
					psgOccupancy,pigResidents,pigLocks,piaQueueB,ihActiveQueueSize,piaQueueA,piDeferredQueueSize,piLockSuccesses);

		} else {
			CUDA_CALL(hipMemset(piaQueueB,0xFF,iQueueSize*sizeof(int)));
			register_deaths<<<hiNumBlocks,NUM_THREADS_PER_BLOCK>>>(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,
					psgOccupancy,pigResidents,pigLocks,piaQueueA,ihActiveQueueSize,piaQueueB,piDeferredQueueSize,piLockSuccesses);
		}
		hipDeviceSynchronize();
		hQueue = !hQueue;
		CUDA_CALL(hipMemcpy(pihDeferredQueueSize,piDeferredQueueSize,sizeof(int),hipMemcpyDeviceToHost));
		printf ("secondary deferrals:%d \n",pihDeferredQueueSize[0]);
	} 

	// for persistent lock failures, use the failsafe version
	if (pihDeferredQueueSize[0] <= 10 || pihDeferredQueueSize[0] >= ihActiveQueueSize) {
		ihActiveQueueSize = pihDeferredQueueSize[0];
		if (hQueue) {
			register_deaths_fs<<<1,1>>>(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,
					psgOccupancy,pigResidents,piaQueueB,ihActiveQueueSize);

		} else {
			register_deaths_fs<<<1,1>>>(psaX,psaY,piaBits,psaAge,pfaSugar,pfaSpice,
					psgOccupancy,pigResidents,piaQueueA,ihActiveQueueSize);
		}
		hipDeviceSynchronize();
	}

	// cleanup
	free(pihLockSuccesses);
	free(pihDeferredQueueSize);
	free(piahTemp);

	return status;
} 
