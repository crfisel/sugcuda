#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "move.h"
#include "die.h"

// this kernel has one thread per agent
__global__ void register_deaths(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, float* pfaSpice, 
		int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, int* piaDeferredQueue, 
		int* piDeferredQueueSize, int* piLockSuccesses)
{
	bool lockFailed = false;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		int iAgentID = piaActiveQueue[iOffset];

		// if agent is alive
		if (psaX[iAgentID] > -1) {

			// check for death by old age or starvation
			// reinterpret piaAgentBits[iAgentID] bitwise
			AgentBitWise abwBits;
			abwBits.asInt = piaAgentBits[iAgentID];

			if ((abwBits.asBits.age > 64+abwBits.asBits.deathAge) || (pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {

				// lock address to register death - if lock fails, defer
				// current agent's address in the grid
				int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];

				// unpack grid bits
				GridBitWise gbwBits;
				gbwBits.asInt = pigGridBits[iAddy];

				// test if square is locked
				if (gbwBits.asBits.isLocked != 0) {
					// if so, lock failed
					lockFailed = true;

				} else {
					// if not, make a copy, but indicating locked
					GridBitWise gbwBitsCopy = gbwBits;
					gbwBitsCopy.asBits.isLocked = 1;

					// now lock the current address if possible
					int iLocked = atomicCAS(&(pigGridBits[iAddy]),gbwBits.asInt,gbwBitsCopy.asInt);
	
					// test if the lock failed
					if (iLocked != gbwBits.asInt) {
						lockFailed = true;
							
					} else {
						// at this point, square is locked and a valid copy of its bits are in gbwBitsCopy (because locked)
							int iFlag = atomicAdd(piLockSuccesses,1);

						// before inserting new resident, check for nonzero occupancy
						if (gbwBitsCopy.asBits.occupancy <= 0) {
									
							// if invalid, unlock with no changes
							iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
								
							// and indicate an error
							printf("underflow occ %d at x:%d y:%d agent %d res %d\n",
								gbwBitsCopy.asBits.occupancy,psaX[iAgentID],psaY[iAgentID],iAgentID,pigResidents[iAddy*MAX_OCCUPANCY]);

						} else {
							// find match starting at end of list
							short k = --gbwBitsCopy.asBits.occupancy;

							// remove current id - if not at the end, replace it with the one from the end and store -1 at end
							if (pigResidents[iAddy*MAX_OCCUPANCY+k] == iAgentID) {
								pigResidents[iAddy*MAX_OCCUPANCY+k] = -1;
							} else {
								while (pigResidents[iAddy*MAX_OCCUPANCY+k] != iAgentID && k >= 0) {k--;}
								if (k != gbwBitsCopy.asBits.occupancy) {
									pigResidents[iAddy*MAX_OCCUPANCY+k] = pigResidents[iAddy*MAX_OCCUPANCY+gbwBitsCopy.asBits.occupancy];
									pigResidents[iAddy*MAX_OCCUPANCY+gbwBitsCopy.asBits.occupancy] = -1;
								}
							}
							
							// TODO: INHERITANCE MUST BE HANDLED BEFORE X POSITION INFO IS ERASED
							// mark agent as dead
							psaX[iAgentID] = SHRT_MIN;
							
							// unlock and update global occupancy values
							gbwBitsCopy.asBits.isLocked = 0;
							iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBitsCopy.asInt);
						}
					}
				}
				// if a death occurred, but lock failures prevented registering it, defer
				if (lockFailed) {
					int iFlag = atomicAdd(piDeferredQueueSize,1);
					piaDeferredQueue[iFlag]=iAgentID;
				}
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
__global__ void register_deaths_fs(short* psaX, short* psaY, int* piaAgentBits, 
		float* pfaSugar, float* pfaSpice, int* pigGridBits, int* pigResidents, 
		int* piaActiveQueue, const int ciActiveQueueSize)
{
	
	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get the iAgentID from the active agent queue
			int iAgentID = piaActiveQueue[iOffset];

			// if agent is alive
			if (psaX[iAgentID] > -1) {

				// reinterpret piaAgentBits bitwise for death age
				AgentBitWise abwBits;
				abwBits.asInt = piaAgentBits[iAgentID];
				// check for death by old age or starvation
				if ((abwBits.asBits.age > 64+(abwBits.asBits.deathAge)) || (pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {
					
					// current agent's address in the grid
					int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
					
					// unpack grid bits
					GridBitWise gbwBits;
					gbwBits.asInt = pigGridBits[iAddy];

					// before removing resident, check for nonzero occupancy
					if (gbwBits.asBits.occupancy <= 0) {
									
						// if invalid, indicate an error
						printf("under occ %d at x:%d y:%d agent %d\n",gbwBits.asBits.occupancy,psaX[iAgentID],psaY[iAgentID],iAgentID);

					} else {
						// find match starting at end of list
						short k = --gbwBits.asBits.occupancy;

						// remove current id - if not at the end, replace it with the one from the end and store -1 at end
						if (pigResidents[iAddy*MAX_OCCUPANCY+k] == iAgentID) {
							pigResidents[iAddy*MAX_OCCUPANCY+k] = -1;
						} else {
							while (pigResidents[iAddy*MAX_OCCUPANCY+k] != iAgentID && k >= 0) {k--;}
							if (k != gbwBits.asBits.occupancy) {
								pigResidents[iAddy*MAX_OCCUPANCY+k] = pigResidents[iAddy*MAX_OCCUPANCY+gbwBits.asBits.occupancy];
								pigResidents[iAddy*MAX_OCCUPANCY+gbwBits.asBits.occupancy] = -1;
							}
						}							

						// TODO: INHERITANCE MUST BE HANDLED BEFORE X POSITION INFO IS ERASED
						// mark agent as dead
						psaX[iAgentID] = SHRT_MIN;
								
						// update global occupancy values
						int iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
					}
				}
			}
		}
	}
	return;
}

