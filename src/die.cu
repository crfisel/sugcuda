#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "move.h"
#include "die.h"

// this kernel has one thread per agent
__global__ void register_deaths(short* psaX, short* psaY, int* piaAgentBits, float* pfaSugar, float* pfaSpice, 
		int* pigGridBits, int* pigResidents, int* piaActiveQueue, const int ciActiveQueueSize, int* piaDeferredQueue, 
		int* piDeferredQueueSize, int* piLockSuccesses)
{
	bool lockFailed = false;

	// get the iAgentID from the active agent queue
	int iOffset = threadIdx.x + blockIdx.x*blockDim.x;
	if (iOffset < ciActiveQueueSize) {
		int iAgentID = piaActiveQueue[iOffset];

		// if agent is alive
		if (psaX[iAgentID] > -1) {

			// check for death by old age or starvation
			// reinterpret piaAgentBits[iAgentID] bitwise
			AgentBitWise abwBits;
			abwBits.asInt = piaAgentBits[iAgentID];

			if ((abwBits.asBits.age > 64+abwBits.asBits.deathAge) || (pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {

				// lock address to register death - if lock fails, defer
				// current agent's address in the grid
				int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];

				// unpack grid bits
				GridBitWise gbwBits;
				gbwBits.asInt = pigGridBits[iAddy];

				// test if square is locked
				if (gbwBits.asBits.isLocked != 0) {
					// if so, lock failed
					lockFailed = true;

				} else {
					// if not, make a copy, but indicating locked
					GridBitWise gbwBitsCopy = gbwBits;
					gbwBitsCopy.asBits.isLocked = 1;

					// now lock the current address if possible
					int iLocked = atomicCAS(&(pigGridBits[iAddy]),gbwBits.asInt,gbwBitsCopy.asInt);
	
					// test if the lock failed
					if (iLocked != gbwBits.asInt) {
						lockFailed = true;
							
					} else {
						// at this point, square is locked and a valid copy of its bits are in gbwBitsCopy (because locked)
							int iFlag = atomicAdd(piLockSuccesses,1);

						// before inserting new resident, check for nonzero occupancy
						if (gbwBitsCopy.asBits.occupancy <= 0) {
									
							// if invalid, unlock with no changes
							iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
								
							// and indicate an error
							printf("underflow occ %d at x:%d y:%d agent %d res %d\n",
								gbwBitsCopy.asBits.occupancy,psaX[iAgentID],psaY[iAgentID],iAgentID,pigResidents[iAddy*MAX_OCCUPANCY]);

						} else {
							remove_resident(&(gbwBitsCopy.asInt),iAddy,pigResidents,iAgentID);

							// mark agent as dead
							psaX[iAgentID] *= -1;
							
							// unlock and update global occupancy values
							gbwBitsCopy.asBits.isLocked = 0;
							iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBitsCopy.asInt);
						}
					}
				}
				// if a death occurred, but lock failures prevented registering it, defer
				if (lockFailed) {
					int iFlag = atomicAdd(piDeferredQueueSize,1);
					piaDeferredQueue[iFlag]=iAgentID;
				}
			}
		}
	}
	return;
}

// this "failsafe" kernel has one thread, for persistent lock failures
__global__ void register_deaths_fs(short* psaX, short* psaY, int* piaAgentBits, 
		float* pfaSugar, float* pfaSpice, int* pigGridBits, int* pigResidents, 
		int* piaActiveQueue, const int ciActiveQueueSize)
{
	
	// only the 1,1 block is active
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		// iterate through the active queue
		for (int iOffset = 0; iOffset < ciActiveQueueSize; iOffset++) {

			// get the iAgentID from the active agent queue
			int iAgentID = piaActiveQueue[iOffset];

			// if agent is alive
			if (psaX[iAgentID] > -1) {

				// reinterpret piaAgentBits bitwise for death age
				AgentBitWise abwBits;
				abwBits.asInt = piaAgentBits[iAgentID];
				// check for death by old age or starvation
				if ((abwBits.asBits.age > 64+(abwBits.asBits.deathAge)) || (pfaSpice[iAgentID] < 0.0f) || (pfaSpice[iAgentID] < 0.0f)) {
					
					// current agent's address in the grid
					int iAddy = psaX[iAgentID]*GRID_SIZE+psaY[iAgentID];
					
					// unpack grid bits
					GridBitWise gbwBits;
					gbwBits.asInt = pigGridBits[iAddy];

					// before removing resident, check for nonzero occupancy
					if (gbwBits.asBits.occupancy <= 0) {
									
						// if invalid, indicate an error
						printf("under occ %d at x:%d y:%d agent %d\n",gbwBits.asBits.occupancy,psaX[iAgentID],psaY[iAgentID],iAgentID);

					} else {
						remove_resident(&(gbwBits.asInt),iAddy,pigResidents,iAgentID);

						// mark agent as dead
						psaX[iAgentID] *= -1;
								
						// update global occupancy values
						int iFlag = atomicExch(&(pigGridBits[iAddy]),gbwBits.asInt);
					}
				}
			}
		}
	}
	return;
}

