#include "hip/hip_runtime.h"
/*
 * harvest.cu
 *
 *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "harvest.h"

__global__ void harvest(unsigned int* pigRandoms, short* psaX, float* pfaSugar, float* pfaSpice,
		int* pigGridBits, int* pigResidents)
{
	short sX = blockIdx.x;
	short sY = threadIdx.x;
	int iAddy = sX*blockDim.x+sY;
	int iAgentID;
	short iOffset;
	GridBitWise gbwBits;

	gbwBits.asInt = pigGridBits[iAddy];
	switch (gbwBits.asBits.occupancy) {
	case 0:
		break;
	case 1:
		iAgentID = pigResidents[iAddy*MAX_OCCUPANCY];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

			pfaSugar[iAgentID] += gbwBits.asBits.sugar;
			pfaSpice[iAgentID] += gbwBits.asBits.spice;
			gbwBits.asBits.sugar = 0;
			gbwBits.asBits.spice = 0;
			pigGridBits[iAddy] = gbwBits.asInt;
		}
		break;
	default:
		float fTemp = pigRandoms[iAddy]*gbwBits.asBits.occupancy/UINT_MAX;
		iOffset = fTemp;
		iAgentID = pigResidents[iAddy*MAX_OCCUPANCY+iOffset];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

			pfaSugar[iAgentID] += gbwBits.asBits.sugar;
			pfaSpice[iAgentID] += gbwBits.asBits.spice;
			gbwBits.asBits.sugar = 0;
			gbwBits.asBits.spice = 0;
			pigGridBits[iAddy] = gbwBits.asInt;
		}
		break;
	}
	return;
}
