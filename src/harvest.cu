#include "hip/hip_runtime.h"
/*
 * harvest.cu
 *
 *  Created on: Dec 3, 2011
 *      Author: C. Richard Fisel
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "symbolic_constants.h"
#include "bitwise.h"
#include "randoms.h"
#include "harvest.h"

__global__ void harvest(unsigned int* theRandoms, short* psaX, float* pfaSugar, float* pfaSpice,
		int* pigGridBits, int* pigResidents)
{
	short sX = blockIdx.x;
	short sY = threadIdx.x;
	int iAddy = sX*blockDim.x+sY;
	int iAgentID;
	short iOffset;
	GridBitWise gbwBits;

	gbwBits.asInt = pigGridBits[iAddy];
	switch (gbwBits.asBits.occupancy) {
	case 0:
		break;
	case 1:
		iAgentID = pigResidents[iAddy*MAX_OCCUPANCY];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

			pfaSugar[iAgentID] += gbwBits.asBits.sugar;
			pfaSpice[iAgentID] += gbwBits.asBits.spice;
		}
		break;
	default:
		float fTemp = theRandoms[iAddy]*gbwBits.asBits.occupancy/UINT_MAX;
		iOffset = fTemp;
		iAgentID = pigResidents[iAddy*MAX_OCCUPANCY+iOffset];

		// if the agent is alive
		if (psaX[iAgentID] > -1) {

			pfaSugar[iAgentID] += gbwBits.asBits.sugar;
			pfaSpice[iAgentID] += gbwBits.asBits.spice;
		}
		break;
	}
	return;
}
